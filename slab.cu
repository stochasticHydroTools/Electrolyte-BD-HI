#include "hip/hip_runtime.h"
/*Raul P.  Pelaez 2020-2023. Dry and  wet diffusion in a slit channel
  with electrostatic interactions.

  About this code
  ------------------

  Besides including many functionalities  from UAMMD, this source also
  includes two headers that are part of this particular project. These
  are:
  1.  RepulsivePotential:  The  definition   of  a  LJ-like  potential
  compatible with UAMMD
  2.  DryDiffusion:  An UAMMD  Integrator  that  mixes UAMMD's  Doubly
  Periodic Stokes  Integrator with  a bare-diffusion  Integrator. This
  allows   to  separate   the  mobility   in  two   parts,  one   with
  hydro. interactions  for the far  field (expensive) and one  for the
  self mobility (fast).

  This source is organized as a series of functions and utility structures.

  Many  of  this  functions   are  called  createSomething.  In  these
  instances, Something refers to some UAMMD-related structure, such as
  the Integrator or the class that takes care of electrostatics.

  The physical domain being simulated is  walled, and as such is prone
  to a particular error where a  particle teleports outside a wall due
  to numerical error.  This source includes a functionality consisting
  of saving the  simulation state every few steps and  rewinding it in
  the  case of  something  like this  happening.  These functions  are
  checkOverlap, saveConfiguration, restoreLastSavedConfiguration,...

  The initialization of  the UAMMD structures and  the main simulation
  loop is carried  out in main().  The length of  this function is due
  to  the implementation  of the  rewind functionality,  which in  its
  current state suffers from a lot of code repetition.

  Basicaly, initialization comes  by calling createIntegrator followed
  by  a series  of calls  to  addInteractor (which  adds a  particular
  interaction to the Integrator) depending  on the options.  Then, the
  simulation is taken  to the next step by calling  forwardTime on the
  Integrator.   Every now  and then  the  state of  the simulation  is
  probed via sim.pd (an instance  of ParticleData, the UAMMD structure
  that holds particle states) and written to disk.

  This code  is GPU enabled  via CUDA, which sometimes  requires doing
  things in  a kind of  convoluted manner. For instance,  the function
  checkOverlap  requires  defining  a structure  called  CheckOverlap,
  whose operator() member  has a fancy __device__  decorator.  Then, a
  thrust  algorithm is  used  to find  possible  overlaps.  You  might
  wonder  why a  simple loop  cannot be  used here.  We could  do that
  following the  rules of CUDA,  but the  boiler plate would  be worse
  and, you'll  have to believe  me, the  resulting code would  be even
  more obtuse  (and inefficient, for  what is worth).

  In CUDA, and in general when writting parallel code in C++, thinking
  in   terms  of   these  "algorithms"   (such  as   copy,  transform,
  find_if,...) is  many times "the  way", in terms  of expressiveness,
  efficiency  and   versatility.   For  instance,  we   can  make  the
  checkOverlap   function   run  in   the   CPU   just  by   replacing
  thrust::cuda::par by thrust::cuda::host. Or the other way around. So
  once you learn to think in  data-driven algorithms, you can get away
  with  not learning  to code  in  a GPU,  since switching  to a  CUDA
  implementation consists of changing a simple word.

*/
#include"uammd.cuh"
#include"RepulsivePotential.cuh"
#include"Interactor/PairForces.cuh"
#include"Interactor/SpectralEwaldPoisson.cuh"
#include"Interactor/ExternalForces.cuh"
#include"Integrator/BrownianDynamics.cuh"
#include"DryDiffusion.cuh"
#include"utils/InputFile.h"
#include"Interactor/DoublyPeriodic/DPPoissonSlab.cuh"
#include<fstream>
#include<limits>
#include<random>
using namespace uammd;
using std::make_shared;
using std::endl;

class RepulsiveWall{
  RepulsivePotentialFunctor::PairParameters params;
  real H;
  real imageDistanceMultiplier = 2.0; //Controls the distance of the image ---->   0  |  0, if this parameter is 2, particles interact with images, if 1, image particles are located at the wall height.
public:
  RepulsiveWall(real H, RepulsivePotentialFunctor::PairParameters ip, real imageDistanceMultiplier):
    H(H),params(ip),imageDistanceMultiplier(imageDistanceMultiplier){}

  __device__ ForceEnergyVirial sum(Interactor::Computables comp, real4 pos /*, real mass */){
    real distanceToImage = abs(abs(pos.z) - H * real(0.5))*imageDistanceMultiplier;
    real fz = RepulsivePotentialFunctor::force(distanceToImage * distanceToImage, params) * distanceToImage;
    ForceEnergyVirial fev;
    fev.force = make_real3(0, 0, fz*(pos.z<0?real(-1.0):real(1.0)));
    return fev;
  }

  auto getArrays(ParticleData *pd){
    auto pos = pd->getPos(access::gpu, access::read);
    return std::make_tuple(pos.raw());
  }
};

class ExternalField{
  real3 externalField;
public:
  ExternalField(real3 externalField):externalField(externalField){}

  __device__ ForceEnergyVirial sum(Interactor::Computables comp, real charges){
    real3 externalForce = charges*externalField;
    ForceEnergyVirial result;
    result.force = externalForce;
    return result;
  }

  auto getArrays(ParticleData *pd){
    auto charges = pd->getCharge(access::gpu, access::read); // a number
    return charges.begin();
  }
};


struct Parameters{
  int numberParticles;
  real Lxy, H;
  int Nxy = -1;
  int support = 10;
  real numberStandardDeviations = 4;
  real upsampling = 1.2;
  real tolerance = 1e-4;
  real temperature;
  real permitivity, permitivityBottom, permitivityTop;

  real bottomWallSurfaceValue = 0;

  int numberSteps, printSteps, relaxSteps;
  real dt, viscosity, hydrodynamicRadius, wetHydrodynamicRadius;

  real gw;
  real U0, sigma, r_m, p, cutOff;
  real wall_U0, wall_sigma, wall_r_m, wall_p, wall_cutOff;
  real imageDistanceMultiplier;

  std::string outfile, readFile, forcefile, fieldfile;
  std::string mobilityFile;


  std::string brownianUpdateRule = "EulerMaruyama";
  bool idealParticles=false;
  bool noElectrostatics=false;
  int w = 6;
  real beta = 10.13641758;
  int nxy_stokes;
  int nz_stokes;
  real hxy_stokes;
  
  real3 externalField;
  int fold;
};

struct UAMMD{
  std::shared_ptr<ParticleData> pd;
  std::shared_ptr<thrust::device_vector<real4>> savedPositions;
  Parameters par;
};

Parameters readParameters(std::string fileName);

void initializeParticles(UAMMD sim){
  auto pos = sim.pd->getPos(access::location::cpu, access::mode::write);
  auto charge = sim.pd->getCharge(access::location::cpu, access::mode::write);
  if(sim.par.readFile.empty()){
    std::generate(pos.begin(), pos.end(),
		  [&](){
		    real Lxy = sim.par.Lxy;
		    real H = sim.par.H;
		    real3 p;
		    real pdf;
		    do{
		      p = make_real3(sim.pd->getSystem()->rng().uniform3(-0.5, 0.5))*make_real3(Lxy, Lxy, H-2*sim.par.gw);
		      pdf = 1.0;
		    }while(sim.pd->getSystem()->rng().uniform(0, 1) > pdf);
		    return make_real4(p, 0);
		  });
    fori(0, sim.par.numberParticles){
      charge[i] = ((i%2)-0.5)*2;
    }
  }
  else{
    std::ifstream in(sim.par.readFile);
    fori(0, sim.par.numberParticles){
      in>>pos[i].x>>pos[i].y>>pos[i].z>>charge[i];
      pos[i].w = 0;
    }
  }
  thrust::copy(pos.begin(), pos.end(), sim.savedPositions->begin());
}

UAMMD initialize(int argc, char *argv[]){
  UAMMD sim;
  auto sys = std::make_shared<System>(argc, argv);
  std::random_device r;
  auto now = static_cast<long long unsigned int>(std::chrono::high_resolution_clock::now().time_since_epoch().count());
  sys->rng().setSeed(now);
  std::string datamain = argc>1?argv[1]:"data.main";
  sim.par = readParameters(datamain);
  sim.pd = std::make_shared<ParticleData>(sim.par.numberParticles, sys);
  sim.savedPositions = std::make_shared<thrust::device_vector<real4>>();
  sim.savedPositions->resize(sim.par.numberParticles);
  initializeParticles(sim);
  return sim;
}

auto string2BrownianRule(std::string str) {
  if(str == "EulerMaruyama") return DryWetBD::update_rules::euler_maruyama;
  else if(str =="Leimkuhler")   return DryWetBD::update_rules::leimkuhler;
  else
    throw std::runtime_error("Invalid brownian rule");
}

auto createIntegrator(UAMMD sim){
  using BD = DryWetBD;
  BD::Parameters par;
  par.temperature = sim.par.temperature;
  par.viscosity = sim.par.viscosity;
  par.hydrodynamicRadius = sim.par.hydrodynamicRadius;
  par.dt = sim.par.dt;
  par.wetRadius = sim.par.wetHydrodynamicRadius;
  par.brownianUpdateRule = string2BrownianRule(sim.par.brownianUpdateRule);
  par.dryMobilityFile = sim.par.mobilityFile;
  par.H = sim.par.H;
  par.Lxy = sim.par.Lxy;
  //par.hxy_stokes = sim.par.hxy_stokes
  // par.w = sim.par.w;
  // par.nxy_stokes = sim.par.nxy_stokes;
  // par.nz_stokes = sim.par.nz_stokes;
  return std::make_shared<BD>(sim.pd, par);
}

auto createDoublyPeriodicElectrostaticInteractor(UAMMD sim){
  DPPoissonSlab::Parameters par;
  par.Lxy = make_real2(sim.par.Lxy);
  par.H = sim.par.H;
  DPPoissonSlab::Permitivity perm;
  perm.inside = sim.par.permitivity;
  perm.top = sim.par.permitivityTop;
  if(sim.par.permitivityTop<0){ //Metallic boundary
    perm.top = std::numeric_limits<real>::infinity();
  }
  perm.bottom = sim.par.permitivityBottom;
  if(sim.par.permitivityBottom<0){ //Metallic boundary
    perm.bottom = std::numeric_limits<real>::infinity();
  }
  par.permitivity = perm;
  par.gw = sim.par.gw;
  par.tolerance = sim.par.tolerance;
  if(sim.par.upsampling > 0){
    par.upsampling=sim.par.upsampling;
  }
  if(sim.par.numberStandardDeviations > 0){
    par.numberStandardDeviations=sim.par.numberStandardDeviations;
  }
  if(sim.par.support > 0){
    par.support=sim.par.support;
  }
  if(sim.par.Nxy > 0){
    par.Nxy = sim.par.Nxy;
  }
  par.support = sim.par.support;
  par.numberStandardDeviations = sim.par.numberStandardDeviations;
  auto dppoisson = std::make_shared<DPPoissonSlab>(sim.pd, par);
  if(sim.par.bottomWallSurfaceValue){
    System::log<System::MESSAGE>("[DPPoisson] Setting the bottom wall zero mode Fourier value to %g", sim.par.bottomWallSurfaceValue);
    dppoisson->setSurfaceValuesZeroModeFourier({0, 0, sim.par.bottomWallSurfaceValue, 0});
  }
  return dppoisson;
}

auto createWallRepulsionInteractor(UAMMD sim){
  RepulsivePotentialFunctor::PairParameters potpar;
  potpar.cutOff2 = sim.par.wall_cutOff*sim.par.wall_cutOff;
  potpar.sigma = sim.par.wall_sigma;
  potpar.U0 = sim.par.wall_U0;
  potpar.r_m = sim.par.wall_r_m;
  potpar.p = sim.par.wall_p;
  return make_shared<ExternalForces<RepulsiveWall>>(sim.pd, make_shared<RepulsiveWall>(sim.par.H, potpar, sim.par.imageDistanceMultiplier));
}

auto createExternalFieldInteractor(UAMMD sim){
  return make_shared<ExternalForces<ExternalField>>(sim.pd, make_shared<ExternalField>(sim.par.externalField));
}

auto createPotential(UAMMD sim){
  auto pot = std::make_shared<RepulsivePotential>();
  RepulsivePotential::InputPairParameters ppar;
  ppar.cutOff = sim.par.cutOff;
  ppar.U0 = sim.par.U0;
  ppar.sigma = sim.par.sigma;
  ppar.r_m = sim.par.r_m;
  ppar.p = sim.par.p;
  System::log<System::MESSAGE>("Repulsive rcut: %g", ppar.cutOff);
  pot->setPotParameters(0, 0, ppar);
 return pot;
}

template<class UsePotential> auto createShortRangeInteractor(UAMMD sim){
  auto pot = createPotential(sim);
  using SR = PairForces<UsePotential>;
  typename SR::Parameters params;
  real Lxy = sim.par.Lxy;
  real H = sim.par.H;
  params.box = Box(make_real3(Lxy, Lxy, H));
  params.box.setPeriodicity(1,1,0);
  auto pairForces = std::make_shared<SR>(sim.pd, params, pot);
  return pairForces;
}

void writeSimulation(UAMMD sim, std::vector<real4> fieldAtParticles){
  auto pos = sim.pd->getPos(access::location::cpu, access::mode::read);
  auto charge = sim.pd->getCharge(access::location::cpu, access::mode::read);
  auto force = sim.pd->getForce(access::location::cpu, access::mode::read);
  static std::ofstream out(sim.par.outfile);
  static std::ofstream outf(sim.par.forcefile);
  static std::ofstream outfield(sim.par.fieldfile);
  Box box(make_real3(sim.par.Lxy, sim.par.Lxy, sim.par.H));
  box.setPeriodicity(1,1,0);
  real3 L = box.boxSize;
  out<<"#Lx="<<L.x*0.5<<";Ly="<<L.y*0.5<<";Lz="<<L.z*0.5<<";"<<std::endl;
  if(outf.good())outf<<"#"<<std::endl;
  if(outfield.good())outfield<<"#"<<std::endl;
  fori(0, sim.par.numberParticles){
    real3 p;
    if(sim.par.fold == 1){
      p = box.apply_pbc(make_real3(pos[i]));
    } else {
      p = make_real3(pos[i]);
    }
    // real3 p = make_real3(pos[i]);
    real q = charge[i];
    out<<std::setprecision(2*sizeof(real))<<p<<" "<<q<<"\n";
    if(outf.good()){
      outf<<std::setprecision(2*sizeof(real))<<force[i]<<"\n";
    }
    if(outfield.good() and fieldAtParticles.size()>0){
      outfield<<std::setprecision(2*sizeof(real))<<fieldAtParticles[i]<<"\n";
    }
  }
  out<<std::flush;
}

struct CheckOverlap {
  real H;
  CheckOverlap(real H):H(H){

  }

  __device__ bool operator()(real4 p){
    return abs(p.z) >= (real(0.5)*H);
  }

};

bool checkWallOverlap(UAMMD sim){
  auto pos = sim.pd->getPos(access::location::gpu, access::mode::read);
  //int overlappingCharges = thrust::count_if(thrust::cuda::par, pos.begin(), pos.end(), CheckOverlap(sim.par.H));
  //return overlappingCharges > 0;
  auto overlappingPos = thrust::find_if(thrust::cuda::par, pos.begin(), pos.end(), CheckOverlap(sim.par.H));
  return overlappingPos != pos.end();
}

void restoreLastSavedConfiguration(UAMMD sim) {
  auto pos = sim.pd->getPos(access::location::gpu, access::mode::write);
  thrust::copy(thrust::cuda::par, sim.savedPositions->begin(), sim.savedPositions->end(), pos.begin());
}

void saveConfiguration(UAMMD sim) {
  auto pos = sim.pd->getPos(access::location::gpu, access::mode::read);
  thrust::copy(thrust::cuda::par, pos.begin(), pos.end(), sim.savedPositions->begin());
}

int main(int argc, char *argv[]){
  auto sim = initialize(argc, argv);
  auto bd = createIntegrator(sim);
  std::shared_ptr<DPPoissonSlab> dpslab;
  if(not sim.par.idealParticles){
    if(not sim.par.noElectrostatics){
      dpslab = createDoublyPeriodicElectrostaticInteractor(sim);
      bd->addInteractor(dpslab);
    }
    if(sim.par.U0 > 0){
      bd->addInteractor(createShortRangeInteractor<RepulsivePotential>(sim));
    }
  }
  bd->addInteractor(createWallRepulsionInteractor(sim));
  if(sim.par.externalField.x != 0 or sim.par.externalField.y != 0 or sim.par.externalField.z != 0){
    bd->addInteractor(createExternalFieldInteractor(sim));
  }
  int numberRetries=0;
  int numberRetriesThisStep=0;
  int lastStepSaved=0;
  constexpr int saveRate = 100;
  constexpr int maximumRetries = 1e6;
  constexpr int maximumRetriesPerStep=1e4;
  forj(0, sim.par.relaxSteps){
    bd->forwardTime();
    if(checkWallOverlap(sim)){
      numberRetries++;
      if(numberRetries>maximumRetries){
	throw std::runtime_error("Too many steps with wall overlapping charges detected, aborting run");
      }
      numberRetriesThisStep++;
      if(numberRetriesThisStep>maximumRetriesPerStep){
	throw std::runtime_error("Cannot recover from configuration with wall overlapping charges, aborting run");
      }
      j=lastStepSaved;
      restoreLastSavedConfiguration(sim);
      continue;
    }
    if(j%saveRate==0){
      numberRetriesThisStep = 0;
      lastStepSaved=j;
      saveConfiguration(sim);
    }

  }
  Timer tim;
  tim.tic();
  lastStepSaved=0;
  forj(0, sim.par.numberSteps){
    bd->forwardTime();
    if(checkWallOverlap(sim)){
      numberRetries++;
      if(numberRetries>maximumRetries){
	throw std::runtime_error("Too many steps with wall overlapping charges detected, aborting run");
      }
      numberRetriesThisStep++;
      if(numberRetriesThisStep>maximumRetriesPerStep){
	throw std::runtime_error("Cannot recover from configuration with wall overlapping charges, aborting run");
      }
      j=lastStepSaved;
      restoreLastSavedConfiguration(sim);
      continue;
    }
    if(j%saveRate==0){
      numberRetriesThisStep=0;
      lastStepSaved=j;
      saveConfiguration(sim);
    }
    if(sim.par.printSteps > 0 and j%sim.par.printSteps==0){
      std::vector<real4> fieldAtParticles;
      if(not sim.par.fieldfile.empty() and dpslab){
	// System::log<System::ERROR>("This functionality is not available");
	auto d_field = dpslab->computeFieldAtParticles();
	fieldAtParticles.resize(d_field.size());
	thrust::copy(d_field.begin(), d_field.end(), fieldAtParticles.begin());
      }
      writeSimulation(sim, fieldAtParticles);
      numberRetriesThisStep=0;
      lastStepSaved=j;
      saveConfiguration(sim);
    }
  }
  System::log<System::MESSAGE>("Number of rejected configurations: %d (%g%% of total)", numberRetries, (double)numberRetries/(sim.par.numberSteps + sim.par.relaxSteps)*100.0);
  auto totalTime = tim.toc();
  System::log<System::MESSAGE>("mean FPS: %.2f", sim.par.numberSteps/totalTime);
  return 0;
}

Parameters readParameters(std::string datamain){
  InputFile in(datamain);
  Parameters par;
  in.getOption("fold", InputFile::Required)>>par.fold;
  in.getOption("Lxy", InputFile::Required)>>par.Lxy;
  in.getOption("H", InputFile::Required)>>par.H;
  in.getOption("numberSteps", InputFile::Required)>>par.numberSteps;
  in.getOption("printSteps", InputFile::Required)>>par.printSteps;
  in.getOption("relaxSteps", InputFile::Required)>>par.relaxSteps;
  in.getOption("dt", InputFile::Required)>>par.dt;
  in.getOption("numberParticles", InputFile::Required)>>par.numberParticles;
  in.getOption("temperature", InputFile::Required)>>par.temperature;
  in.getOption("viscosity", InputFile::Required)>>par.viscosity;
  in.getOption("hydrodynamicRadius", InputFile::Required)>>par.hydrodynamicRadius;
  in.getOption("outfile", InputFile::Required)>>par.outfile;
  in.getOption("useMobilityFromFile", InputFile::Optional)>>par.mobilityFile;
  in.getOption("forcefile", InputFile::Optional)>>par.forcefile;
  in.getOption("fieldfile", InputFile::Optional)>>par.fieldfile;
  in.getOption("U0", InputFile::Required)>>par.U0;
  in.getOption("r_m", InputFile::Required)>>par.r_m;
  in.getOption("p", InputFile::Required)>>par.p;
  in.getOption("sigma", InputFile::Required)>>par.sigma;
  in.getOption("readFile", InputFile::Optional)>>par.readFile;
  in.getOption("wetHydrodynamicRadius", InputFile::Required)>>par.wetHydrodynamicRadius;
  in.getOption("gw", InputFile::Required)>>par.gw;
  in.getOption("tolerance", InputFile::Optional)>>par.tolerance;
  in.getOption("permitivity", InputFile::Required)>>par.permitivity;
  in.getOption("permitivityTop", InputFile::Required)>>par.permitivityTop;
  in.getOption("permitivityBottom", InputFile::Required)>>par.permitivityBottom;
  in.getOption("externalField", InputFile::Required)>>par.externalField;

  in.getOption("Nxy", InputFile::Required)>>par.Nxy;

  in.getOption("wall_U0", InputFile::Required)>>par.wall_U0;
  in.getOption("wall_r_m", InputFile::Required)>>par.wall_r_m;
  in.getOption("wall_p", InputFile::Required)>>par.wall_p;
  in.getOption("wall_sigma", InputFile::Required)>>par.wall_sigma;
  in.getOption("imageDistanceMultiplier", InputFile::Required)>>par.imageDistanceMultiplier;
  par.wall_cutOff = par.wall_sigma*pow(2,1.0/par.wall_p);
  par.cutOff = par.sigma*pow(2,1.0/par.p);
  in.getOption("BrownianUpdateRule", InputFile::Optional)>>par.brownianUpdateRule;
  if(in.getOption("idealParticles", InputFile::Optional))
    par.idealParticles = true;
  if(in.getOption("noElectrostatics", InputFile::Optional))
    par.noElectrostatics = true;

  in.getOption("bottomWallSurfaceValue", InputFile::Optional)>>par.bottomWallSurfaceValue;

  // in.getOption("hxy_stokes", InputFile::Required)>>par.hxy_stokes;
  // in.getOption("w", InputFile::Required)>>par.w;
  // in.getOption("beta", InputFile::Required)>>par.beta;
  // in.getOption("nxy_stokes", InputFile::Required)>>par.nxy_stokes;
  // in.getOption("nz_stokes", InputFile::Required)>>par.nz_stokes;

  return par;
}
