#include "hip/hip_runtime.h"
/* Raul P. Pelaez and Aref Hashemi 2023. Tests for the dry wet slit channel Integrator

 */
#include <gtest/gtest.h>
#include "gmock/gmock.h"
#include <random>
#include <fstream>
#include "DryDiffusion.cuh"
#include "Interactor/DoublyPeriodic/DPPoissonSlab.cuh"
#include "uammd/src/utils/complex.cuh"
#include <thrust/device_vector.h>
#include <random>
#include "/usr/include/eigen3/Eigen/Dense"
using namespace uammd;

using scalar = double;

// A normalized measure for accuracy
bool tolerance(scalar val, scalar expectedval, scalar numdigits = 1.9){
  // std::cout << val << " " << expectedval << std::endl;
  if (val == expectedval){
    return true;
  } else {
    if (abs(expectedval) > 1e-15){
      scalar numCorrectDigits = log10(abs(expectedval/(val-expectedval)));
      if (numCorrectDigits > numdigits){
	return true;
      } else {
	return false;
      }
    } else {
      if (-log10(abs(val-expectedval))>14){
	return true;
      } else {
	return false;
      }
    }
  }

}

//Writes a mobility file with constant mobility accross the domain
void writeDefaultMobilityFile(){
  std::ofstream out("uniformMob.dat");
  out<<"-1.0 1.0 1.0 1.0"<<std::endl;
  out<<"0.0 1.0 1.0 1.0"<<std::endl;
  out<<"1.0 1.0 1.0 1.0"<<std::endl;
}

struct Parameters{
  int numberParticles;
  real Lxy, H;
  int Nxy = -1;
  int support = 10;
  real numberStandardDeviations = 4;
  real upsampling = 1.2;
  real tolerance = 1e-4;
  real temperature;
  real permitivity, permitivityBottom, permitivityTop;

  real bottomWallSurfaceValue = 0;

  int numberSteps, printSteps, relaxSteps;
  real dt, viscosity, hydrodynamicRadius, wetHydrodynamicRadius, wetFraction;

  real gw;
  real U0, sigma, r_m, p, cutOff;
  real wall_U0, wall_sigma, wall_r_m, wall_p, wall_cutOff;
  real imageDistanceMultiplier;

  std::string outfile, readFile, forcefile, fieldfile;
  std::string mobilityFile;

  std::string brownianUpdateRule = "EulerMaruyama";
  bool idealParticles=false;
  bool noElectrostatics=false;
  int w = 6;
  real beta = 10.13641758;
  int nxy_stokes;
  int nz_stokes;
  real hxy_stokes;

  real3 externalField;
  int fold;
};


struct UAMMD{
  std::shared_ptr<ParticleData> pd;
  std::shared_ptr<thrust::device_vector<real4>> savedPositions;
  Parameters par;
};

//Adds a force to the first particle, the force defaults to 1,0,0 if not provided
struct miniInteractor: public Interactor{
  real3 f;
public:
  miniInteractor(std::shared_ptr<ParticleData> pd, real3 f = {1,0,0}):
    Interactor(pd), f(f){
  }

  void sum(Computables comp, hipStream_t st =0) override{
    pd->getForce(access::cpu, access::write)[0] = make_real4(f);
  }
};

// ############## Tests by Aref ############## //

// Printing average velocity velocity (in the xy plane) as a function of z
TEST(FLUIDVELOCITY,CanPrint){
  DPStokesSlab_ns::DPStokes::Parameters par;
  par.viscosity = 1.0/(6*M_PI);
  par.hydrodynamicRadius = 1;
  par.Lx = 76.8;
  par.H = 19.2;
  real hxy_stokes = 0.64;

  auto dppar = getDPStokesParamtersOnlyForce(par.Lx, par.H, par.viscosity, par.hydrodynamicRadius, hxy_stokes);
  auto dpstokes = std::make_shared<DPStokesSlab_ns::DPStokes>(dppar);

  real z = 0;
  auto pos = thrust::make_constant_iterator<real4>({0,0,z,0});
  auto force = thrust::make_constant_iterator<real4>({1,0,0,0});
  auto MdotOut = dpstokes->Mdot(pos, force, 1);
  real muxx = 6*M_PI*par.viscosity*par.hydrodynamicRadius*real3(MdotOut[0]).x;
  std::cout << "mobility = " << muxx << std::endl;

  std::vector<double> averageVelocity = dpstokes->computeAverageVelocity(pos, force, 1, 0);// 0 denotes x direction
  for (int i=0;i<averageVelocity.size();i++){
    std::cout << averageVelocity[i] << std::endl;
  }
  // a dummy check!
  double a = 1;
  EXPECT_THAT(a, ::testing::DoubleNear(1, 1e-5));
}

// validation of the average velocity versus continuum
// Many particles are randomly placed in the simulation box with an ad-hoc force (in the x direction) on each of them that correspond to a force density; uammd solutions will be compared to the analytical solution to the 1D Stokes equation. Data will be written in 'fluidVelocity.dat' as z, uammd vel, analytical vel.
TEST(FLUIDVELOCITY,Validation){
  DPStokesSlab_ns::DPStokes::Parameters par;
  par.viscosity = 1.0/(6*M_PI);
  par.hydrodynamicRadius = 1;
  real hxy_stokes = 0.64*par.hydrodynamicRadius;
  par.Lx = 40;
  par.H = 10;

  auto dppar = getDPStokesParamtersOnlyForce(par.Lx, par.H, par.viscosity, par.hydrodynamicRadius, hxy_stokes);
  auto dpstokes = std::make_shared<DPStokesSlab_ns::DPStokes>(dppar);

  real forceDensity = 1;//f_x in the Stokes equarion: \mu\frac{d^2u}{dz^2}=-f_x
  real packFactor = 10;
  int numberParticles = (int)(pow(par.Lx,2)*par.H/pow(hxy_stokes,3)*packFactor);
  std::cout << "number of farticles: " << numberParticles << std::endl;
  real forceParticle = forceDensity*forceDensity*pow(par.Lx,2)*par.H/((real)numberParticles);
  real loz = -0.5*par.H, rangez = par.H;
  real loxy = 0, rangexy = par.Lx;
  real x, y, z;
  srand((unsigned) time(NULL));
  auto pd = std::make_shared<ParticleData>(numberParticles);
  std::ofstream randPosOut("randomPos.dat");
  for (int i=0;i<numberParticles;i++){
    z = loz+(real)rand()/(real)RAND_MAX*rangez;
    x = loxy+(real)rand()/(real)RAND_MAX*rangexy;
    y = loxy+(real)rand()/(real)RAND_MAX*rangexy;
    randPosOut << x << " " << y << " " << z << std::endl;
    pd->getPos(access::gpu, access::write)[i] = {x,y,z,0};
    pd->getForce(access::gpu, access::write)[i] = {forceParticle,0,0,0};
  }
  randPosOut.close();

  auto pos = pd->getPos(access::gpu, access::read);
  auto force = pd->getForce(access::gpu, access::read);
  
  std::vector<double> averageVelocity = dpstokes->computeAverageVelocity(pos, force, numberParticles, 0);
  std::ofstream out("fluidVelocity.dat");
  // Note Chebyshev points are (b+a)/2+(b-a)/2*cos(j*pi/(nz-1)) for j = 0,1,...,nz-1
  real a = -0.5*par.H, b = 0.5*par.H;
  int nz = averageVelocity.size();
  std::vector<double> averageVelocityContinuum(nz);// solution from solving the continuum Stokes sequation with a force density: u = fx*H^2/(8*mu)*(1-4(z/H)^2)
  real zpos;
  real err = 0;
  for (int i=0;i<nz;i++){
    zpos = (b+a)/2+(b-a)/2*cos(i*M_PI/(nz-1));
    averageVelocityContinuum[i] = forceDensity*pow(par.H,2)/(8*par.viscosity)*(1-4*pow(zpos/par.H,2));
    out << zpos/b << " " << averageVelocity[i] << " " << averageVelocityContinuum[i] << std::endl;
    err += averageVelocityContinuum[i]!=0?pow(abs((averageVelocityContinuum[i]-averageVelocity[i])/averageVelocityContinuum[i]),2):abs(averageVelocityContinuum[i]-averageVelocity[i]);
  }
  out.close();
  // One can also plot the data to observe how the two solutions compare.
  std::cout << "mean squared error = " << err/nz << std::endl; 
  real tol = 0.01;
  EXPECT_THAT(err/nz, ::testing::Le(tol));
}

// Reading and writing particle positions
TEST(Playing, ReadWriteParticlePosition){
  auto pd = std::make_shared<ParticleData>(2);
  pd->getPos(access::location::cpu, access::mode::write)[0] = {1,0,0};
  pd->getPos(access::location::cpu, access::mode::write)[1] = {1.5,0,0};
  real x0 = pd->getPos(access::location::cpu, access::mode::read)[0].x;
  real x1 = pd->getPos(access::location::cpu, access::mode::read)[1].x;
  std::cout << "particle #1 position = " << x0 << std::endl;
  std::cout << "particle #2 position = " << x1 << std::endl;
  EXPECT_THAT(x0, ::testing::DoubleNear(1, 1e-5));
  EXPECT_THAT(x1, ::testing::DoubleNear(1.5, 1e-5));
}

// Full Dry Mode: Compute electrostatic fields for 2 particles
// The computed electric fields at particles will be compared to that obtained from the MATLAB code.
TEST(FULLDRY, ComputeElectrostaticField){
  using DPP = DPPoissonSlab;
  DPP::Parameters par;
  // needs a real2 for Lxy (see DPPoissonSlab.cuh)
  par.Lxy = {76.8, 76.8};
  par.H = 19.2;
  par.gw = 0.25;
  DPP::Permitivity perm;
  perm.inside = 1.0;
  perm.top = 0.05;
  perm.bottom = 0.05;
  par.permitivity = perm;
  par.Nxy = 72;
  
  auto pd = std::make_shared<ParticleData>(2);
  pd->getPos(access::location::cpu, access::mode::write)[0] = {7,6,-4.8};
  pd->getPos(access::location::cpu, access::mode::write)[1] = {11,6,-4.8};
  pd->getCharge(access::location::cpu, access::mode::write)[0] = 1;
  pd->getCharge(access::location::cpu, access::mode::write)[1] = -1;

  real c0 = pd->getCharge(access::location::cpu, access::mode::read)[0];
  std::cout << "charge of particle #1 = " << c0 << std::endl;
  real c1 = pd->getCharge(access::location::cpu, access::mode::read)[1];
  std::cout << "charge of particle #2 = " << c1 << std::endl;
  
  auto poisson = std::make_shared<DPPoissonSlab>(pd, par);
  thrust::device_vector<real4> field = poisson->computeFieldAtParticles();
  std::vector<real4> fieldAtParticles;
  fieldAtParticles.resize(field.size());
  thrust::copy(field.begin(), field.end(), fieldAtParticles.begin());
  real E0x = fieldAtParticles[0].x;
  real E0y = fieldAtParticles[0].y;
  real E0z = fieldAtParticles[0].z;
  std::cout << "x field at particle #1 (electrostatics) = " << E0x << std::endl;
  std::cout << "y field at particle #1 (electrostatics) = " << E0y << std::endl;
  std::cout << "z field at particle #1 (electrostatics) = " << E0z << std::endl;
  real expectedE0x = 0.005240820721856;//given by the MATLB code
  real expectedE0y = 0                ;
  real expectedE0z = 0.000164511031114;
  EXPECT_THAT(tolerance(E0x, expectedE0x, 4), ::testing::IsTrue);
  EXPECT_THAT(tolerance(E0y, expectedE0y, 4), ::testing::IsTrue);
  EXPECT_THAT(tolerance(E0z, expectedE0z, 4), ::testing::IsTrue);
}

// Full Dry Mode: Test if a simple integration is working properly
// An external force [1 0 0] is applied on a particle and we compute its displacement when mobility is 1.
TEST(FULLDRY, SimpleIntegration){
  using BD = DryWetBD;
  BD::Parameters par;
  writeDefaultMobilityFile();
  par.dryMobilityFile = "uniformMob.dat";
  par.temperature = 0;
  // \mu = 1/(6\pi\eta a) = 1
  par.viscosity = 1.0/(6*M_PI);
  par.hydrodynamicRadius = 1.0;
  par.dt = 1.0;
  par.wetFraction = 0;
  par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  par.H = 19.2;
  par.Lxy = 76.8;
  auto pd = std::make_shared<ParticleData>(1);
  pd->getPos(access::cpu, access::write)[0] = {0,0,0};
  auto bd = std::make_shared<BD>(pd, par);
  bd->addInteractor(std::make_shared<miniInteractor>(pd));
  bd->forwardTime();
  // F = [1, 0, 0]; \mu = 1; U = 1; \Delta t = 1. \Rightarrow \Delta x = 1
  real dx = pd->getPos(access::cpu, access::write)[0].x;
  EXPECT_THAT(dx, ::testing::DoubleNear(1, 1e-5));
}

// Full Dry Mode: Test if an integration works for a pair of particles interacting electrostatically
// Here we assume that the self mobility is given by the Einstein equation.
TEST(FULLDRY, IntegrationFlatMobility){
  UAMMD sim;
  
  sim.par.numberParticles = 2;
  sim.pd = std::make_shared<ParticleData>(sim.par.numberParticles);
  
  sim.pd->getPos(access::location::cpu, access::mode::write)[0] = {7,6,-4.8};
  sim.pd->getPos(access::location::cpu, access::mode::write)[1] = {11,6,-4.8};
  sim.pd->getCharge(access::location::cpu, access::mode::write)[0] = 1;
  sim.pd->getCharge(access::location::cpu, access::mode::write)[1] = -1;

  // {
  //   auto pos = sim.pd->getPos(access::cpu, access::write);
  //   pos[0] = {1,2,3};
  //   pos[1] = {2,3,4};
  // }
  
  sim.par.Lxy = 76.8;
  sim.par.H = 19.2;
  sim.par.gw = 0.25;
  sim.par.permitivity = 1;
  sim.par.permitivityTop = 0.05;
  sim.par.permitivityBottom = 0.05;
  sim.par.Nxy = 72;
  sim.par.temperature = 0;
  sim.par.viscosity = 1.0/(6*M_PI);
  sim.par.hydrodynamicRadius = 1.0;
  sim.par.wetFraction = 0;
  sim.par.dt = 0.01;
  sim.par.brownianUpdateRule = "EulerMaruyama";

  using DPP = DPPoissonSlab;
  DPP::Parameters par;
  par.Lxy = make_real2(sim.par.Lxy);
  par.H = sim.par.H;
  par.gw = sim.par.gw;
  DPP::Permitivity perm;
  perm.inside = sim.par.permitivity;
  perm.top = sim.par.permitivityTop;
  perm.bottom = sim.par.permitivityBottom;
  par.permitivity = perm;
  par.Nxy = sim.par.Nxy;
  auto poisson = std::make_shared<DPPoissonSlab>(sim.pd, par);
  thrust::device_vector<real4> field = poisson->computeFieldAtParticles();
  std::vector<real4> fieldAtParticles;
  fieldAtParticles.resize(field.size());
  thrust::copy(field.begin(), field.end(), fieldAtParticles.begin());
  scalar F0x = fieldAtParticles[0].x;
  scalar F0y = fieldAtParticles[0].y;
  scalar F0z = fieldAtParticles[0].z;
  
  using BD = DryWetBD;
  BD::Parameters parBD;
  writeDefaultMobilityFile();
  parBD.dryMobilityFile = "uniformMob.dat";
  parBD.temperature = sim.par.temperature;
  parBD.viscosity = sim.par.viscosity;
  parBD.hydrodynamicRadius = sim.par.hydrodynamicRadius;
  parBD.wetFraction = sim.par.wetFraction;
  parBD.dt = sim.par.dt;
  parBD.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  parBD.Lxy = sim.par.Lxy;
  parBD.H = sim.par.H;
  auto bd = std::make_shared<BD>(sim.pd, parBD);
  bd->addInteractor(poisson);
  bd->forwardTime();

  scalar dx = sim.pd->getPos(access::cpu, access::write)[0].x-7;
  scalar dy = sim.pd->getPos(access::cpu, access::write)[0].y-6;
  scalar dz = sim.pd->getPos(access::cpu, access::write)[0].z+4.8;
  std::cout << "x displacement is " << dx << std::endl;
  std::cout << "y displacement is " << dy << std::endl;
  std::cout << "z displacement is " << dz << std::endl;
  scalar expecteddx = F0x*1*sim.par.dt;//0.0000524082072186 given by the MATLB code  
  scalar expecteddy = F0y*1*sim.par.dt;//0
  scalar expecteddz = F0z*1*sim.par.dt;//0.0000016451103111;
  EXPECT_THAT(tolerance(dx, expecteddx, 4), ::testing::IsTrue);
  EXPECT_THAT(tolerance(dy, expecteddy, 4), ::testing::IsTrue);
  EXPECT_THAT(tolerance(dz, expecteddz, 4), ::testing::IsTrue);
}


// Test if the self mobility is computed correctly
TEST(FULLDRY, selfMobility){
  auto pd = std::make_shared<ParticleData>(1);//dummy instance
  using BD = DryWetBD;
  BD::Parameters parBD;
  parBD.viscosity = 1.0/(6*M_PI);
  parBD.hydrodynamicRadius = 1;
  parBD.Lxy = 76.8;
  parBD.H = 19.2;
  parBD.hxy_stokes = 0.64;
  parBD.wetFraction = 0;
  auto bd = std::make_shared<BD>(pd, parBD);

  real z = 4;
  real4 mobilityData = computeSelfMobility(parBD, z);
  scalar muxx = mobilityData.y;
  scalar muyy = mobilityData.z;
  scalar muzz = mobilityData.w;
  std::cout << muxx << std::endl;
  std::cout << muyy << std::endl;
  std::cout << muzz << std::endl;
  scalar expectedmuxx = 0.849327755959755;// computed by the DPStokes python code at z = 4R_h above the bottom wall
  scalar expectedmuyy = 0.849291424072988;
  scalar expectedmuzz = 0.724165665045591;
  std::cout << "expected xx mobility is " << expectedmuxx << std::endl;
  std::cout << "expected yy mobility is " << expectedmuyy << std::endl;
  std::cout << "expected zz mobility is " << expectedmuzz << std::endl;
  EXPECT_THAT(tolerance(muxx, expectedmuxx), ::testing::IsTrue);
  EXPECT_THAT(tolerance(muyy, expectedmuyy), ::testing::IsTrue);
  EXPECT_THAT(tolerance(muzz, expectedmuzz), ::testing::IsTrue);
}


// Full Dry Mode: Test if an integration works for a pair of particles interacting electrostatically
// Here self mobility is precomputed by the Stokes solver.
TEST(FULLDRY, Integration){
  UAMMD sim;
  
  sim.par.numberParticles = 2;
  sim.pd = std::make_shared<ParticleData>(sim.par.numberParticles);
  
  sim.par.Lxy = 76.8;
  sim.par.H = 19.2;
  scalar yp0  = 0.5*sim.par.Lxy;
  scalar yp1  = yp0;
  scalar zp0 = -0.5*sim.par.H+4;
  scalar zp1 = -0.5*sim.par.H+7;
  scalar xp0 = 0.5*sim.par.Lxy+2;
  scalar xp1 = 0.5*sim.par.Lxy-2;
  
  sim.pd->getPos(access::location::cpu, access::mode::write)[0] = {xp0,yp0,zp0};
  sim.pd->getPos(access::location::cpu, access::mode::write)[1] = {xp1,yp1,zp1};
  sim.pd->getCharge(access::location::cpu, access::mode::write)[0] = 1;
  sim.pd->getCharge(access::location::cpu, access::mode::write)[1] = -1;

  sim.par.gw = 0.25;
  sim.par.permitivity = 1;
  sim.par.permitivityTop = 0.05;
  sim.par.permitivityBottom = 0.05;
  sim.par.Nxy = 72;
  sim.par.temperature = 0;
  sim.par.viscosity = 1.0/(6*M_PI);
  sim.par.hydrodynamicRadius = 1.0;
  sim.par.wetFraction = 0;
  sim.par.dt = 0.01;
  sim.par.brownianUpdateRule = "EulerMaruyama";

  using DPP = DPPoissonSlab;
  DPP::Parameters par;
  par.Lxy = make_real2(sim.par.Lxy);
  par.H = sim.par.H;
  par.gw = sim.par.gw;
  DPP::Permitivity perm;
  perm.inside = sim.par.permitivity;
  perm.top = sim.par.permitivityTop;
  perm.bottom = sim.par.permitivityBottom;
  par.permitivity = perm;
  par.Nxy = sim.par.Nxy;
  auto poisson = std::make_shared<DPPoissonSlab>(sim.pd, par);
  thrust::device_vector<real4> field = poisson->computeFieldAtParticles();
  std::vector<real4> fieldAtParticles;
  fieldAtParticles.resize(field.size());
  thrust::copy(field.begin(), field.end(), fieldAtParticles.begin());
  scalar F0x = fieldAtParticles[0].x;
  scalar F0y = fieldAtParticles[0].y;
  scalar F0z = fieldAtParticles[0].z;

  
  using BD = DryWetBD;
  BD::Parameters parBD;
  // writeDefaultMobilityFile();
  // parBD.dryMobilityFile = "uniformMob.dat";
  parBD.temperature = sim.par.temperature;
  parBD.viscosity = sim.par.viscosity;
  parBD.hydrodynamicRadius = sim.par.hydrodynamicRadius;
  parBD.wetFraction = sim.par.wetFraction;
  parBD.dt = sim.par.dt;
  parBD.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  parBD.Lxy = sim.par.Lxy;
  parBD.H = sim.par.H;
  parBD.hxy_stokes = 0.64;
  auto bd = std::make_shared<BD>(sim.pd, parBD);
  bd->addInteractor(poisson);
  bd->forwardTime();
  
  scalar dx = sim.pd->getPos(access::cpu, access::write)[0].x-xp0;
  scalar dy = sim.pd->getPos(access::cpu, access::write)[0].y-yp0;
  scalar dz = sim.pd->getPos(access::cpu, access::write)[0].z-zp0;
  std::ofstream out("FULLDRYdisplacements.dat");// Will be needed for the next test (periodicity)
  out << dx << " " << dy << " " << dz << std::endl;
  std::cout << "x displacement is " << dx << std::endl;
  std::cout << "y displacement is " << dy << std::endl;
  std::cout << "z displacement is " << dz << std::endl;
  scalar mu_xx = 0.849327755959755;// computed by the DPStokes python code at z = 4R_h above the bottom wall
  scalar mu_yy = 0.849291424072988;
  scalar mu_zz = 0.724165665045591;
  scalar expecteddx = mu_xx*F0x*sim.par.dt;
  scalar expecteddy = mu_yy*F0y*sim.par.dt;
  scalar expecteddz = mu_zz*F0z*sim.par.dt;
  std::cout << "expected x displacement is " << expecteddx << std::endl;
  std::cout << "expected y displacement is " << expecteddy << std::endl;
  std::cout << "expected z displacement is " << expecteddz << std::endl;
  EXPECT_THAT(tolerance(dx, expecteddx), ::testing::IsTrue);
  EXPECT_THAT(tolerance(dy, expecteddy), ::testing::IsTrue);
  EXPECT_THAT(tolerance(dz, expecteddz), ::testing::IsTrue);
}


// Full Dry Mode: periodicity
// This test follows the previous test.
TEST(FULLDRY, Periodicity){
  UAMMD sim;
  
  sim.par.numberParticles = 2;
  sim.pd = std::make_shared<ParticleData>(sim.par.numberParticles);
  
  sim.par.Lxy = 76.8;
  sim.par.H = 19.2;
  scalar yp0  = 0.5*sim.par.Lxy;
  scalar yp1  = yp0;
  scalar zp0 = -0.5*sim.par.H+4;
  scalar zp1 = -0.5*sim.par.H+7;
  scalar xp0 = 1*sim.par.Lxy+2;// Moved to the next block
  scalar xp1 = 1*sim.par.Lxy-2;
  
  sim.pd->getPos(access::location::cpu, access::mode::write)[0] = {xp0,yp0,zp0};
  sim.pd->getPos(access::location::cpu, access::mode::write)[1] = {xp1,yp1,zp1};
  sim.pd->getCharge(access::location::cpu, access::mode::write)[0] = 1;
  sim.pd->getCharge(access::location::cpu, access::mode::write)[1] = -1;

  sim.par.gw = 0.25;
  sim.par.permitivity = 1;
  sim.par.permitivityTop = 0.05;
  sim.par.permitivityBottom = 0.05;
  sim.par.Nxy = 72;
  sim.par.temperature = 0;
  sim.par.viscosity = 1.0/(6*M_PI);
  sim.par.hydrodynamicRadius = 1.0;
  sim.par.wetFraction = 0;
  sim.par.dt = 0.01;
  sim.par.brownianUpdateRule = "EulerMaruyama";

  using DPP = DPPoissonSlab;
  DPP::Parameters par;
  par.Lxy = make_real2(sim.par.Lxy);
  par.H = sim.par.H;
  par.gw = sim.par.gw;
  DPP::Permitivity perm;
  perm.inside = sim.par.permitivity;
  perm.top = sim.par.permitivityTop;
  perm.bottom = sim.par.permitivityBottom;
  par.permitivity = perm;
  par.Nxy = sim.par.Nxy;
  auto poisson = std::make_shared<DPPoissonSlab>(sim.pd, par);
  thrust::device_vector<real4> field = poisson->computeFieldAtParticles();
  std::vector<real4> fieldAtParticles;
  fieldAtParticles.resize(field.size());
  thrust::copy(field.begin(), field.end(), fieldAtParticles.begin());
  scalar F0x = fieldAtParticles[0].x;
  scalar F0y = fieldAtParticles[0].y;
  scalar F0z = fieldAtParticles[0].z;
  
  using BD = DryWetBD;
  BD::Parameters parBD;
  // writeDefaultMobilityFile();
  // parBD.dryMobilityFile = "uniformMob.dat";
  parBD.temperature = sim.par.temperature;
  parBD.viscosity = sim.par.viscosity;
  parBD.hydrodynamicRadius = sim.par.hydrodynamicRadius;
  parBD.wetFraction = sim.par.wetFraction;
  parBD.dt = sim.par.dt;
  parBD.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  parBD.Lxy = sim.par.Lxy;
  parBD.H = sim.par.H;
  parBD.hxy_stokes = 0.64;
  auto bd = std::make_shared<BD>(sim.pd, parBD);
  bd->addInteractor(poisson);
  bd->forwardTime();
  
  scalar dx = sim.pd->getPos(access::cpu, access::write)[0].x-xp0;
  scalar dy = sim.pd->getPos(access::cpu, access::write)[0].y-yp0;
  scalar dz = sim.pd->getPos(access::cpu, access::write)[0].z-zp0;
  std::ifstream in("FULLDRYdisplacements.dat");// Should be available from the previous test
  double s;
  std::vector<double> expectedvals;
  while(in >> s){
    expectedvals.push_back(s);
  }
  
  std::cout << "x displacement is " << dx << std::endl;
  std::cout << "y displacement is " << dy << std::endl;
  std::cout << "z displacement is " << dz << std::endl;
  
  EXPECT_THAT(tolerance(dx, expectedvals[0]), ::testing::IsTrue);
  EXPECT_THAT(tolerance(dy, expectedvals[1]), ::testing::IsTrue);
  EXPECT_THAT(tolerance(dz, expectedvals[2]), ::testing::IsTrue);
}

// Full Wet Mode: Test if an integration works for a pair of particles
TEST(FULLWET, Integration){
  UAMMD sim;
  
  sim.par.numberParticles = 2;
  sim.pd = std::make_shared<ParticleData>(sim.par.numberParticles);
  sim.par.Lxy = 76.8;
  sim.par.H = 19.2;
  scalar yp0  = 0.5*sim.par.Lxy;
  scalar yp1  = yp0;
  scalar zp0 = -0.5*sim.par.H+4;
  scalar zp1 = -0.5*sim.par.H+7;
  scalar xp0 = 0.5*sim.par.Lxy+2;
  scalar xp1 = 0.5*sim.par.Lxy-2;
  
  sim.pd->getPos(access::location::cpu, access::mode::write)[0] = {xp0,yp0,zp0};
  sim.pd->getPos(access::location::cpu, access::mode::write)[1] = {xp1,yp1,zp1};
  sim.pd->getCharge(access::location::cpu, access::mode::write)[0] = 1;
  sim.pd->getCharge(access::location::cpu, access::mode::write)[1] = -1;

  real c0 = sim.pd->getCharge(access::location::cpu, access::mode::read)[0];
  real c1 = sim.pd->getCharge(access::location::cpu, access::mode::read)[1];
   
  sim.par.gw = 0.25;
  sim.par.permitivity = 1;
  sim.par.permitivityTop = 0.05;
  sim.par.permitivityBottom = 0.05;
  sim.par.Nxy = 135;
  sim.par.temperature = 0;
  sim.par.viscosity = 1.0/(6*M_PI);
  sim.par.hydrodynamicRadius = 1.0;
  sim.par.wetFraction = 1;
  sim.par.dt = 1;
  sim.par.brownianUpdateRule = "EulerMaruyama";

  using DPP = DPPoissonSlab;
  DPP::Parameters par;
  par.Lxy = make_real2(sim.par.Lxy);
  par.H = sim.par.H;
  par.gw = sim.par.gw;
  DPP::Permitivity perm;
  perm.inside = sim.par.permitivity;
  perm.top = sim.par.permitivityTop;
  perm.bottom = sim.par.permitivityBottom;
  par.permitivity = perm;
  par.Nxy = sim.par.Nxy;
  auto poisson = std::make_shared<DPPoissonSlab>(sim.pd, par);
  thrust::device_vector<real4> field = poisson->computeFieldAtParticles();
  std::vector<real4> fieldAtParticles;
  fieldAtParticles.resize(field.size());
  thrust::copy(field.begin(), field.end(), fieldAtParticles.begin());
  scalar F0x = c0*fieldAtParticles[0].x;
  scalar F0y = c0*fieldAtParticles[0].y;
  scalar F0z = c0*fieldAtParticles[0].z;
  scalar F1x = c1*fieldAtParticles[1].x;
  scalar F1y = c1*fieldAtParticles[1].y;
  scalar F1z = c1*fieldAtParticles[1].z;
  std::cout << "F0x = " << F0x << std::endl;
  std::cout << "F0y = " << F0y << std::endl;
  std::cout << "F0z = " << F0z << std::endl;
  std::cout << "F1x = " << F1x << std::endl;
  std::cout << "F1y = " << F1y << std::endl;
  std::cout << "F1z = " << F1z << std::endl;
  
  using BD = DryWetBD;
  BD::Parameters parBD;
  parBD.temperature = sim.par.temperature;
  parBD.viscosity = sim.par.viscosity;
  parBD.hydrodynamicRadius = sim.par.hydrodynamicRadius;
  parBD.wetFraction = sim.par.wetFraction;
  parBD.dt = sim.par.dt;
  parBD.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  parBD.Lxy = sim.par.Lxy;
  parBD.H = sim.par.H;
  parBD.hxy_stokes = 0.64;
  auto bd = std::make_shared<BD>(sim.pd, parBD);
  bd->addInteractor(poisson);
  bd->forwardTime();

  scalar dx = sim.pd->getPos(access::cpu, access::write)[1].x-xp1;
  scalar dy = sim.pd->getPos(access::cpu, access::write)[1].y-yp1;
  scalar dz = sim.pd->getPos(access::cpu, access::write)[1].z-zp1;
  std::ofstream out("FULLWETdisplacements.dat");// Will be needed for the next test (periodicity)
  out << dx << " " << dy << " " << dz << std::endl;
  std::cout << "x displacement is " << dx << std::endl;
  std::cout << "y displacement is " << dy << std::endl;
  std::cout << "z displacement is " << dz << std::endl;
  
  //given by the DPStokes solver code (pair_mobility.py)
  scalar nu_xx = 0.138291767457352;
  scalar nu_xy = 0;
  scalar nu_xz = -0.038706105653531;
  scalar nu_yx = 0;
  scalar nu_yy = 0.047304644896516;
  scalar nu_yz = 0;
  scalar nu_zx = -0.081720265183282;
  scalar nu_zy = 0;
  scalar nu_zz = 0.037554341201021;
  scalar mu_xx = 0.893993039433745;
  scalar mu_yy = 0.893956879655325;
  scalar mu_zz = 0.829029667444467;
  scalar expecteddx = (nu_xx*F0x+nu_xy*F0y+nu_xz*F0z+mu_xx*F1x)*sim.par.dt;
  scalar expecteddy = (nu_yx*F0x+nu_yy*F0y+nu_yz*F0z+mu_yy*F1y)*sim.par.dt;
  scalar expecteddz = (nu_zx*F0x+nu_zy*F0y+nu_zz*F0z+mu_zz*F1z)*sim.par.dt;
  std::cout << "expected x displacement is " << expecteddx << std::endl;
  std::cout << "expected y displacement is " << expecteddy << std::endl;
  std::cout << "expected z displacement is " << expecteddz << std::endl;
  EXPECT_THAT(tolerance(dx, expecteddx), ::testing::IsTrue);
  EXPECT_THAT(tolerance(dy, expecteddy), ::testing::IsTrue);
  EXPECT_THAT(tolerance(dz, expecteddz), ::testing::IsTrue);
}

// Full Wet Mode: periodicity
// This test follows the previous test.
TEST(FULLWET, Periodicity){
  UAMMD sim;
  
  sim.par.numberParticles = 2;
  sim.pd = std::make_shared<ParticleData>(sim.par.numberParticles);
  sim.par.Lxy = 76.8;
  sim.par.H = 19.2;
  scalar yp0  = 0.5*sim.par.Lxy;
  scalar yp1  = yp0;
  scalar zp0 = -0.5*sim.par.H+4;
  scalar zp1 = -0.5*sim.par.H+7;
  scalar xp0 = 1*sim.par.Lxy+2;// Moved to the next block
  scalar xp1 = 1*sim.par.Lxy-2;
  
  sim.pd->getPos(access::location::cpu, access::mode::write)[0] = {xp0,yp0,zp0};
  sim.pd->getPos(access::location::cpu, access::mode::write)[1] = {xp1,yp1,zp1};
  sim.pd->getCharge(access::location::cpu, access::mode::write)[0] = 1;
  sim.pd->getCharge(access::location::cpu, access::mode::write)[1] = -1;

  real c0 = sim.pd->getCharge(access::location::cpu, access::mode::read)[0];
  real c1 = sim.pd->getCharge(access::location::cpu, access::mode::read)[1];
   
  sim.par.gw = 0.25;
  sim.par.permitivity = 1;
  sim.par.permitivityTop = 0.05;
  sim.par.permitivityBottom = 0.05;
  sim.par.Nxy = 135;
  sim.par.temperature = 0;
  sim.par.viscosity = 1.0/(6*M_PI);
  sim.par.hydrodynamicRadius = 1.0;
  sim.par.wetFraction = 1;
  sim.par.dt = 1;
  sim.par.brownianUpdateRule = "EulerMaruyama";

  using DPP = DPPoissonSlab;
  DPP::Parameters par;
  par.Lxy = make_real2(sim.par.Lxy);
  par.H = sim.par.H;
  par.gw = sim.par.gw;
  DPP::Permitivity perm;
  perm.inside = sim.par.permitivity;
  perm.top = sim.par.permitivityTop;
  perm.bottom = sim.par.permitivityBottom;
  par.permitivity = perm;
  par.Nxy = sim.par.Nxy;
  auto poisson = std::make_shared<DPPoissonSlab>(sim.pd, par);
  thrust::device_vector<real4> field = poisson->computeFieldAtParticles();
  std::vector<real4> fieldAtParticles;
  fieldAtParticles.resize(field.size());
  thrust::copy(field.begin(), field.end(), fieldAtParticles.begin());
  scalar F0x = c0*fieldAtParticles[0].x;
  scalar F0y = c0*fieldAtParticles[0].y;
  scalar F0z = c0*fieldAtParticles[0].z;
  scalar F1x = c1*fieldAtParticles[1].x;
  scalar F1y = c1*fieldAtParticles[1].y;
  scalar F1z = c1*fieldAtParticles[1].z;
  std::cout << "F0x = " << F0x << std::endl;
  std::cout << "F0y = " << F0y << std::endl;
  std::cout << "F0z = " << F0z << std::endl;
  std::cout << "F1x = " << F1x << std::endl;
  std::cout << "F1y = " << F1y << std::endl;
  std::cout << "F1z = " << F1z << std::endl;
  
  using BD = DryWetBD;
  BD::Parameters parBD;
  parBD.temperature = sim.par.temperature;
  parBD.viscosity = sim.par.viscosity;
  parBD.hydrodynamicRadius = sim.par.hydrodynamicRadius;
  parBD.wetFraction = sim.par.wetFraction;
  parBD.dt = sim.par.dt;
  parBD.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  parBD.Lxy = sim.par.Lxy;
  parBD.H = sim.par.H;
  parBD.hxy_stokes = 0.64;
  auto bd = std::make_shared<BD>(sim.pd, parBD);
  bd->addInteractor(poisson);
  bd->forwardTime();

  scalar dx = sim.pd->getPos(access::cpu, access::write)[1].x-xp1;
  scalar dy = sim.pd->getPos(access::cpu, access::write)[1].y-yp1;
  scalar dz = sim.pd->getPos(access::cpu, access::write)[1].z-zp1;
  std::ifstream in("FULLWETdisplacements.dat");// Should be available from the previous test
  double s;
  std::vector<double> expectedvals;
  while(in >> s){
    expectedvals.push_back(s);
  }
  
  std::cout << "x displacement is " << dx << std::endl;
  std::cout << "y displacement is " << dy << std::endl;
  std::cout << "z displacement is " << dz << std::endl;
  
  EXPECT_THAT(tolerance(dx, expectedvals[0]), ::testing::IsTrue);
  EXPECT_THAT(tolerance(dy, expectedvals[1]), ::testing::IsTrue);
  EXPECT_THAT(tolerance(dz, expectedvals[2]), ::testing::IsTrue);
}


// 50% Wet, 50% Dry: Test if an integration works for a pair of particles
TEST(DRYWET, Integration){
  using Eigen::MatrixXd;
  using Eigen::VectorXd;
  UAMMD sim;
  
  sim.par.numberParticles = 2;
  sim.pd = std::make_shared<ParticleData>(sim.par.numberParticles);
  sim.par.Lxy = 76.8;
  sim.par.H = 19.2;
  scalar yp0  = 0.5*sim.par.Lxy;
  scalar yp1  = yp0;
  scalar zp0 = -0.5*sim.par.H+4;
  scalar zp1 = -0.5*sim.par.H+7;
  scalar xp0 = 0.5*sim.par.Lxy+2;
  scalar xp1 = 0.5*sim.par.Lxy-2;
  
  sim.pd->getPos(access::location::cpu, access::mode::write)[0] = {xp0,yp0,zp0};
  sim.pd->getPos(access::location::cpu, access::mode::write)[1] = {xp1,yp1,zp1};
  sim.pd->getCharge(access::location::cpu, access::mode::write)[0] = 1;
  sim.pd->getCharge(access::location::cpu, access::mode::write)[1] = -1;

  real c0 = sim.pd->getCharge(access::location::cpu, access::mode::read)[0];
  real c1 = sim.pd->getCharge(access::location::cpu, access::mode::read)[1];
   
  sim.par.gw = 0.25;
  sim.par.permitivity = 1;
  sim.par.permitivityTop = 0.05;
  sim.par.permitivityBottom = 0.05;
  sim.par.Nxy = 135;
  sim.par.temperature = 0;
  sim.par.viscosity = 1.0/(6*M_PI);
  sim.par.hydrodynamicRadius = 1.0;
  sim.par.wetFraction = 0.5;
  sim.par.dt = 1;
  sim.par.brownianUpdateRule = "EulerMaruyama";

  using DPP = DPPoissonSlab;
  DPP::Parameters par;
  par.Lxy = make_real2(sim.par.Lxy);
  par.H = sim.par.H;
  par.gw = sim.par.gw;
  DPP::Permitivity perm;
  perm.inside = sim.par.permitivity;
  perm.top = sim.par.permitivityTop;
  perm.bottom = sim.par.permitivityBottom;
  par.permitivity = perm;
  par.Nxy = sim.par.Nxy;
  auto poisson = std::make_shared<DPPoissonSlab>(sim.pd, par);
  thrust::device_vector<real4> field = poisson->computeFieldAtParticles();
  std::vector<real4> fieldAtParticles;
  fieldAtParticles.resize(field.size());
  thrust::copy(field.begin(), field.end(), fieldAtParticles.begin());

  VectorXd F(6);
  F << c0*fieldAtParticles[0].x, c0*fieldAtParticles[0].y, c0*fieldAtParticles[0].z,
       c1*fieldAtParticles[1].x, c1*fieldAtParticles[1].y, c1*fieldAtParticles[1].z;
  
  using BD = DryWetBD;
  BD::Parameters parBD;
  parBD.temperature = sim.par.temperature;
  parBD.viscosity = sim.par.viscosity;
  parBD.hydrodynamicRadius = sim.par.hydrodynamicRadius;
  parBD.wetFraction = sim.par.wetFraction;
  parBD.dt = sim.par.dt;
  parBD.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  parBD.Lxy = sim.par.Lxy;
  parBD.H = sim.par.H;
  parBD.hxy_stokes = 1.28;
  auto bd = std::make_shared<BD>(sim.pd, parBD);
  bd->addInteractor(poisson);
  bd->forwardTime();

  scalar dx = sim.pd->getPos(access::cpu, access::write)[1].x-xp1;
  scalar dy = sim.pd->getPos(access::cpu, access::write)[1].y-yp1;
  scalar dz = sim.pd->getPos(access::cpu, access::write)[1].z-zp1;
  std::cout << "x displacement is " << dx << std::endl;
  std::cout << "y displacement is " << dy << std::endl;
  std::cout << "z displacement is " << dz << std::endl;

  // mobility matrix computed by the DPStokes solver (python interface)
  MatrixXd mWet(6,6);
  mWet.row(0) << 0.708884362108862, 0, 0, 0.257442239887106, 0, -0.126448728742101;
  mWet.row(1) << 0, 0.708894111245968, 0, 0, 0.120242666875890, 0;
  mWet.row(2) << 0, 0, 0.489019735544522, -0.051068001579525, 0, 0.081803173383461;
  mWet.row(3) << 0.257442991316777, 0, -0.051065753076216, 0.791052032682486, 0, 0;
  mWet.row(4) << 0, 0.120242407250183, 0, 0, 0.791061727610104, 0;
  mWet.row(5) << -0.126448405742272, 0, 0.081803264694953, 0, 0, 0.666364258639729;
  std::cout << "M_wet = " << std::endl;
  std::cout << mWet << std::endl;

  MatrixXd mDry(6,6);
  mDry.row(0) << 0.708884362108862, 0, 0, 0, 0, 0;
  mDry.row(1) << 0, 0.708894111245968, 0, 0, 0, 0;
  mDry.row(2) << 0, 0, 0.489019735544522, 0, 0, 0;
  mDry.row(3) << 0, 0, 0, 0.791052032682486, 0, 0;
  mDry.row(4) << 0, 0, 0, 0, 0.791061727610104, 0;
  mDry.row(5) << 0, 0, 0, 0, 0, 0.666364258639729;
  std::cout << "M_dry = " << std::endl;
  std::cout << mDry << std::endl;

  VectorXd displacement(6);
  displacement = 0.5*(mWet+mDry)*F*sim.par.dt;

  scalar expecteddx = displacement(3);
  scalar expecteddy = displacement(4);
  scalar expecteddz = displacement(5);
  std::cout << "expected x displacement is " << expecteddx << std::endl;
  std::cout << "expected y displacement is " << expecteddy << std::endl;
  std::cout << "expected z displacement is " << expecteddz << std::endl;
  EXPECT_THAT(tolerance(dx, expecteddx), ::testing::IsTrue);
  EXPECT_THAT(tolerance(dy, expecteddy), ::testing::IsTrue);
  EXPECT_THAT(tolerance(dz, expecteddz), ::testing::IsTrue);
}

// ############## Tests by Raul ############## //
TEST(DryWetMobility, CanBeCreated){
  using BD = DryWetBD;
  BD::Parameters par;
  par.temperature = 1.0;
  par.viscosity = 1.0;
  par.hydrodynamicRadius = 1.0;
  par.dt = 1.0;
  par.wetFraction = 0.7;
  par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  par.H = 16;
  par.Lxy = 32;
  par.hxy_stokes = -1.0;
  auto pd = std::make_shared<ParticleData>(1);
  auto bd = std::make_shared<BD>(pd, par);
}


TEST(FullDryMobility, SelfMobilityIsCorrect){
  using BD = DryWetBD;
  writeDefaultMobilityFile();
  BD::Parameters par;
  par.temperature = 0;
  par.viscosity = 1.0/(6*M_PI);
  par.hydrodynamicRadius = 1.0;
  par.dt = 1.0;
  par.wetFraction = 0;
  par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  par.dryMobilityFile = "uniformMob.dat";
  par.H = 32;
  par.Lxy = 64;
  par.hxy_stokes = -1.0;
  auto pd = std::make_shared<ParticleData>(1);
  pd->getPos(access::cpu, access::write)[0] = real4();
  auto bd = std::make_shared<BD>(pd, par);
  bd->addInteractor(std::make_shared<miniInteractor>(pd));
  bd->forwardTime();
  real M0 = pd->getPos(access::cpu, access::write)[0].x;
  ASSERT_THAT(M0, ::testing::DoubleNear(1, 1e-5));
}

TEST(FullWetMobility, SelfMobilityIsCorrectAtMiddlePlaneForLargeDomain){
  using BD = DryWetBD;
  writeDefaultMobilityFile();
  BD::Parameters par;
  par.temperature = 0;
  par.viscosity = 1.0/(6*M_PI);
  par.hydrodynamicRadius = 1.0;
  par.dt = 1.0;
  par.wetFraction = 1;
  par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  par.dryMobilityFile = "uniformMob.dat";
  par.H = 128;
  par.Lxy = 64;
  par.hxy_stokes = -1.0;
  auto pd = std::make_shared<ParticleData>(1);
  pd->getPos(access::cpu, access::write)[0] = real4();
  auto bd = std::make_shared<BD>(pd, par);
  bd->addInteractor(std::make_shared<miniInteractor>(pd));
  bd->forwardTime();
  real M0 = pd->getPos(access::cpu, access::write)[0].x;
  ASSERT_THAT(M0, ::testing::DoubleNear(1, 1e-1));
}


//Asserts the correctness of the self mobility for a certain wet radius
//All other parameters are hardcoded, see the function.
//The total hydrodynamic radius is 1
void computeSelfMobilityWithWetRadius(real wetFraction){
  using BD = DryWetBD;
  writeDefaultMobilityFile();
  BD::Parameters par;
  par.temperature = 0;
  par.viscosity = 1.0/(6*M_PI);
  par.hydrodynamicRadius = 1.0;
  par.dt = 1.0;
  par.wetFraction = wetFraction;
  par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  par.dryMobilityFile = "uniformMob.dat";
  par.H = 64;
  par.Lxy = 64;
  par.hxy_stokes = -1.0;
  auto pd = std::make_shared<ParticleData>(1);
  pd->getPos(access::cpu, access::write)[0] = real4();
  auto bd = std::make_shared<BD>(pd, par);
  bd->addInteractor(std::make_shared<miniInteractor>(pd));
  bd->forwardTime();
  real M0 = pd->getPos(access::cpu, access::write)[0].x;
  ASSERT_THAT(M0, ::testing::DoubleNear(1, 1e-1))<<"Failed with wet fraction "<<wetFraction;
}

TEST(DryWetMobility, SelfMobilityIsCorrectForAnyWetRadius){
  real minwetFraction = 0;
  real maxwetFraction = 1;
  int Ntest = 4;
  fori(0, Ntest){
    real wetFraction = minwetFraction + i*(maxwetFraction - minwetFraction)/(Ntest-1);
    computeSelfMobilityWithWetRadius(wetFraction);
  }
}

