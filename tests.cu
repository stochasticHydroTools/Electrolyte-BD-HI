#include "hip/hip_runtime.h"
/* Raul P. Pelaez and Aref Hashemi 2023. Tests for the dry wet slit channel Integrator

 */
#include <gtest/gtest.h>
#include "gmock/gmock.h"
#include <random>
#include <fstream>
#include "DryDiffusion.cuh"
#include"Interactor/DoublyPeriodic/DPPoissonSlab.cuh"
using namespace uammd;

using scalar = double;

// A normalized measure for accuracy
bool tolerance(scalar val, scalar expectedval, scalar numdigits = 2.0){
  std::cout << val << " " << expectedval << std::endl;
  if (abs(expectedval) > 1e-15){
    scalar numCorrectDigits = log10(abs(expectedval/(val-expectedval)));
    if (numCorrectDigits > numdigits){
      return true;
    } else {
      return false;
    }
  } else {
    if (-log10(abs(val-expectedval))>14){
      return true;
    } else {
      return false;
    }
  }

}

//Writes a mobility file with constant mobility accross the domain
void writeDefaultMobilityFile(){
  std::ofstream out("uniformMob.dat");
  out<<"-1.0 1.0 1.0 1.0"<<std::endl;
  out<<"0.0 1.0 1.0 1.0"<<std::endl;
  out<<"1.0 1.0 1.0 1.0"<<std::endl;
}

struct Parameters{
  int numberParticles;
  real Lxy, H;
  int Nxy = -1;
  int support = 10;
  real numberStandardDeviations = 4;
  real upsampling = 1.2;
  real tolerance = 1e-4;
  real temperature;
  real permitivity, permitivityBottom, permitivityTop;

  real bottomWallSurfaceValue = 0;

  int numberSteps, printSteps, relaxSteps;
  real dt, viscosity, hydrodynamicRadius, wetHydrodynamicRadius;

  real gw;
  real U0, sigma, r_m, p, cutOff;
  real wall_U0, wall_sigma, wall_r_m, wall_p, wall_cutOff;
  real imageDistanceMultiplier;

  std::string outfile, readFile, forcefile, fieldfile;
  std::string mobilityFile;


  std::string brownianUpdateRule = "EulerMaruyama";
  bool idealParticles=false;
  bool noElectrostatics=false;
  int w = 6;
  real beta = 10.13641758;
  int nxy_stokes;
  int nz_stokes;

  real3 externalField;
  int fold;
};


struct UAMMD{
  std::shared_ptr<ParticleData> pd;
  std::shared_ptr<thrust::device_vector<real4>> savedPositions;
  Parameters par;
};

//Adds a force to the first particle, the force defaults to 1,0,0 if not provided
struct miniInteractor: public Interactor{
  real3 f;
public:
  miniInteractor(std::shared_ptr<ParticleData> pd, real3 f = {1,0,0}):
    Interactor(pd), f(f){
  }

  void sum(Computables comp, hipStream_t st =0) override{
    pd->getForce(access::cpu, access::write)[0] = make_real4(f);
  }
};

// ############## Tests by Aref ############## //

// Add a force to a particle and recompute it using the Intercator
TEST(Playing, ReadWriteParticleForce){
  auto pd = std::make_shared<ParticleData>(2);
  auto external = std::make_shared<miniInteractor>(pd);
  external->sum({.force=true});
  real3 F0 = make_real3(pd->getForce(access::location::cpu, access::mode::read)[0]);
  std::cout << "force on particle #1 = " << F0 << std::endl;
  EXPECT_THAT(F0.x, ::testing::DoubleNear(1, 1e-5));
}

// Reading and writing particle positions
TEST(Playing, ReadWriteParticlePosition){
  auto pd = std::make_shared<ParticleData>(2);
  pd->getPos(access::location::cpu, access::mode::write)[0] = {1,0,0};
  pd->getPos(access::location::cpu, access::mode::write)[1] = {1.5,0,0};
  real x0 = pd->getPos(access::location::cpu, access::mode::read)[0].x;
  real x1 = pd->getPos(access::location::cpu, access::mode::read)[1].x;
  std::cout << "particle #1 position = " << x0 << std::endl;
  std::cout << "particle #2 position = " << x1 << std::endl;
  EXPECT_THAT(x0, ::testing::DoubleNear(1, 1e-5));
  EXPECT_THAT(x1, ::testing::DoubleNear(1.5, 1e-5));
}

// Full Dry Mode: Compute electrostatic fields for 2 particles
// The computed electric fields at particles will be compared to that obtained from the MATLAB code.
TEST(FULLDRY, ComputeElectrostaticField){
  using DPP = DPPoissonSlab;
  DPP::Parameters par;
  // needs a real2 for Lxy (see DPPoissonSlab.cuh)
  par.Lxy = {76.8, 76.8};
  par.H = 19.2;
  par.gw = 0.25;
  DPP::Permitivity perm;
  perm.inside = 1.0;
  perm.top = 0.05;
  perm.bottom = 0.05;
  par.permitivity = perm;
  par.Nxy = 72;
  
  auto pd = std::make_shared<ParticleData>(2);
  pd->getPos(access::location::cpu, access::mode::write)[0] = {7,6,-4.8};
  pd->getPos(access::location::cpu, access::mode::write)[1] = {11,6,-4.8};
  pd->getCharge(access::location::cpu, access::mode::write)[0] = 1;
  pd->getCharge(access::location::cpu, access::mode::write)[1] = -1;

  real c0 = pd->getCharge(access::location::cpu, access::mode::read)[0];
  std::cout << "charge of particle #1 = " << c0 << std::endl;
  real c1 = pd->getCharge(access::location::cpu, access::mode::read)[1];
  std::cout << "charge of particle #2 = " << c1 << std::endl;
  
  auto poisson = std::make_shared<DPPoissonSlab>(pd, par);
  thrust::device_vector<real4> field = poisson->computeFieldAtParticles();
  std::vector<real4> fieldAtParticles;
  fieldAtParticles.resize(field.size());
  thrust::copy(field.begin(), field.end(), fieldAtParticles.begin());
  real E0x = fieldAtParticles[0].x;
  real E0y = fieldAtParticles[0].y;
  real E0z = fieldAtParticles[0].z;
  std::cout << "x field at particle #1 (electrostatics) = " << E0x << std::endl;
  std::cout << "y field at particle #1 (electrostatics) = " << E0y << std::endl;
  std::cout << "z field at particle #1 (electrostatics) = " << E0z << std::endl;
  real expectedE0x = 0.005240820721856;//given by the MATLB code
  real expectedE0y = 0                ;
  real expectedE0z = 0.000164511031114;
  EXPECT_THAT(tolerance(E0x, expectedE0x, 4), ::testing::IsTrue);
  EXPECT_THAT(tolerance(E0y, expectedE0y, 4), ::testing::IsTrue);
  EXPECT_THAT(tolerance(E0z, expectedE0z, 4), ::testing::IsTrue);
}

// Full Dry Mode: Test if a simple integration is working properly
// An external force [1 0 0] is applied on a particle and we compute its displacement when mobility is 1.
TEST(FULLDRY, SimpleIntegration){
  using BD = DryWetBD;
  BD::Parameters par;
  writeDefaultMobilityFile();
  par.dryMobilityFile = "uniformMob.dat";
  par.temperature = 0;
  // \mu = 1/(6\pi\eta a) = 1
  par.viscosity = 1.0/(6*M_PI);
  par.hydrodynamicRadius = 1.0;
  par.dt = 1.0;
  par.wetRadius = -1;
  par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  par.H = 19.2;
  par.Lxy = 76.8;
  auto pd = std::make_shared<ParticleData>(1);
  pd->getPos(access::cpu, access::write)[0] = {0,0,0};
  auto bd = std::make_shared<BD>(pd, par);
  bd->addInteractor(std::make_shared<miniInteractor>(pd));
  bd->forwardTime();
  // F = [1, 0, 0]; \mu = 1; U = 1; \Delta t = 1. \Rightarrow \Delta x = 1
  real dx = pd->getPos(access::cpu, access::write)[0].x;
  EXPECT_THAT(dx, ::testing::DoubleNear(1, 1e-5));
}

// Full Dry Mode: Test if an integration works for a pair of particles interacting electrostatically
// Here we assume that the self mobility is given by the Einstein equation.
TEST(FULLDRY, IntegrationFlatMobility){
  UAMMD sim;
  
  sim.par.numberParticles = 2;
  sim.pd = std::make_shared<ParticleData>(sim.par.numberParticles);
  
  sim.pd->getPos(access::location::cpu, access::mode::write)[0] = {7,6,-4.8};
  sim.pd->getPos(access::location::cpu, access::mode::write)[1] = {11,6,-4.8};
  sim.pd->getCharge(access::location::cpu, access::mode::write)[0] = 1;
  sim.pd->getCharge(access::location::cpu, access::mode::write)[1] = -1;

  // {
  //   auto pos = sim.pd->getPos(access::cpu, access::write);
  //   pos[0] = {1,2,3};
  //   pos[1] = {2,3,4};
  // }
  
  sim.par.Lxy = 76.8;
  sim.par.H = 19.2;
  sim.par.gw = 0.25;
  sim.par.permitivity = 1;
  sim.par.permitivityTop = 0.05;
  sim.par.permitivityBottom = 0.05;
  sim.par.Nxy = 72;
  sim.par.temperature = 0;
  sim.par.viscosity = 1.0/(6*M_PI);
  sim.par.hydrodynamicRadius = 1.0;
  sim.par.wetHydrodynamicRadius = -1;
  sim.par.dt = 0.01;
  sim.par.brownianUpdateRule = "EulerMaruyama";

  using DPP = DPPoissonSlab;
  DPP::Parameters par;
  par.Lxy = make_real2(sim.par.Lxy);
  par.H = sim.par.H;
  par.gw = sim.par.gw;
  DPP::Permitivity perm;
  perm.inside = sim.par.permitivity;
  perm.top = sim.par.permitivityTop;
  perm.bottom = sim.par.permitivityBottom;
  par.permitivity = perm;
  par.Nxy = sim.par.Nxy;
  auto poisson = std::make_shared<DPPoissonSlab>(sim.pd, par);
  thrust::device_vector<real4> field = poisson->computeFieldAtParticles();
  std::vector<real4> fieldAtParticles;
  fieldAtParticles.resize(field.size());
  thrust::copy(field.begin(), field.end(), fieldAtParticles.begin());
  scalar F0x = fieldAtParticles[0].x;
  scalar F0y = fieldAtParticles[0].y;
  scalar F0z = fieldAtParticles[0].z;
  
  using BD = DryWetBD;
  BD::Parameters parBD;
  writeDefaultMobilityFile();
  parBD.dryMobilityFile = "uniformMob.dat";
  parBD.temperature = sim.par.temperature;
  parBD.viscosity = sim.par.viscosity;
  parBD.hydrodynamicRadius = sim.par.hydrodynamicRadius;
  parBD.wetRadius = sim.par.wetHydrodynamicRadius;
  parBD.dt = sim.par.dt;
  parBD.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  parBD.Lxy = sim.par.Lxy;
  parBD.H = sim.par.H;
  auto bd = std::make_shared<BD>(sim.pd, parBD);
  bd->addInteractor(poisson);
  bd->forwardTime();

  scalar dx = sim.pd->getPos(access::cpu, access::write)[0].x-7;
  scalar dy = sim.pd->getPos(access::cpu, access::write)[0].y-6;
  scalar dz = sim.pd->getPos(access::cpu, access::write)[0].z+4.8;
  std::cout << "x displacement is " << dx << std::endl;
  std::cout << "y displacement is " << dy << std::endl;
  std::cout << "z displacement is " << dz << std::endl;
  scalar expecteddx = F0x*1*sim.par.dt;//0.0000524082072186 given by the MATLB code  
  scalar expecteddy = F0y*1*sim.par.dt;//0
  scalar expecteddz = F0z*1*sim.par.dt;//0.0000016451103111;
  EXPECT_THAT(tolerance(dx, expecteddx, 4), ::testing::IsTrue);
  EXPECT_THAT(tolerance(dy, expecteddy, 4), ::testing::IsTrue);
  EXPECT_THAT(tolerance(dz, expecteddz, 4), ::testing::IsTrue);
}


// Test if the self mobility is computed correctly
TEST(FULLDRY, selfMobility){
  auto pd = std::make_shared<ParticleData>(1);//dummy instance
  using BD = DryWetBD;
  BD::Parameters parBD;
  parBD.viscosity = 1.0/(6*M_PI);
  parBD.hydrodynamicRadius = 1;
  parBD.Lxy = 76.8;
  parBD.H = 19.2;
  auto bd = std::make_shared<BD>(pd, parBD);

  real z = 4;
  real4 mobilityData = computeSelfMobility(parBD, z);
  scalar muxx = mobilityData.y;
  scalar muyy = mobilityData.z;
  scalar muzz = mobilityData.w;
  std::cout << muxx << std::endl;
  std::cout << muyy << std::endl;
  std::cout << muzz << std::endl;
  scalar expectedmuxx = 0.849327755959755;// computed by the DPStokes python code at z = 4R_h above the bottom wall
  scalar expectedmuyy = 0.849291424072988;
  scalar expectedmuzz = 0.724165665045591;
  std::cout << "expected xx mobility is " << expectedmuxx << std::endl;
  std::cout << "expected yy mobility is " << expectedmuyy << std::endl;
  std::cout << "expected zz mobility is " << expectedmuzz << std::endl;
  EXPECT_THAT(tolerance(muxx, expectedmuxx), ::testing::IsTrue);
  EXPECT_THAT(tolerance(muyy, expectedmuyy), ::testing::IsTrue);
  EXPECT_THAT(tolerance(muzz, expectedmuzz), ::testing::IsTrue);
}


// Full Dry Mode: Test if an integration works for a pair of particles interacting electrostatically
// Here self mobility is precomputed by the Stokes solver.
TEST(FULLDRY, Integration){
  UAMMD sim;
  
  sim.par.numberParticles = 2;
  sim.pd = std::make_shared<ParticleData>(sim.par.numberParticles);
  
  sim.par.Lxy = 76.8;
  sim.par.H = 19.2;
  scalar yp0  = 0.5*sim.par.Lxy;
  scalar yp1  = yp0;
  scalar zp0 = -0.5*sim.par.H+4;
  scalar zp1 = -0.5*sim.par.H+7;
  scalar xp0 = 0.5*sim.par.Lxy+2;
  scalar xp1 = 0.5*sim.par.Lxy-2;
  
  sim.pd->getPos(access::location::cpu, access::mode::write)[0] = {xp0,yp0,zp0};
  sim.pd->getPos(access::location::cpu, access::mode::write)[1] = {xp1,yp1,zp1};
  sim.pd->getCharge(access::location::cpu, access::mode::write)[0] = 1;
  sim.pd->getCharge(access::location::cpu, access::mode::write)[1] = -1;

  sim.par.gw = 0.25;
  sim.par.permitivity = 1;
  sim.par.permitivityTop = 0.05;
  sim.par.permitivityBottom = 0.05;
  sim.par.Nxy = 72;
  sim.par.temperature = 0;
  sim.par.viscosity = 1.0/(6*M_PI);
  sim.par.hydrodynamicRadius = 1.0;
  sim.par.wetHydrodynamicRadius = -1;
  sim.par.dt = 0.01;
  sim.par.brownianUpdateRule = "EulerMaruyama";

  using DPP = DPPoissonSlab;
  DPP::Parameters par;
  par.Lxy = make_real2(sim.par.Lxy);
  par.H = sim.par.H;
  par.gw = sim.par.gw;
  DPP::Permitivity perm;
  perm.inside = sim.par.permitivity;
  perm.top = sim.par.permitivityTop;
  perm.bottom = sim.par.permitivityBottom;
  par.permitivity = perm;
  par.Nxy = sim.par.Nxy;
  auto poisson = std::make_shared<DPPoissonSlab>(sim.pd, par);
  thrust::device_vector<real4> field = poisson->computeFieldAtParticles();
  std::vector<real4> fieldAtParticles;
  fieldAtParticles.resize(field.size());
  thrust::copy(field.begin(), field.end(), fieldAtParticles.begin());
  scalar F0x = fieldAtParticles[0].x;
  scalar F0y = fieldAtParticles[0].y;
  scalar F0z = fieldAtParticles[0].z;

  
  using BD = DryWetBD;
  BD::Parameters parBD;
  // writeDefaultMobilityFile();
  // parBD.dryMobilityFile = "uniformMob.dat";
  parBD.temperature = sim.par.temperature;
  parBD.viscosity = sim.par.viscosity;
  parBD.hydrodynamicRadius = sim.par.hydrodynamicRadius;
  parBD.wetRadius = sim.par.wetHydrodynamicRadius;
  parBD.dt = sim.par.dt;
  parBD.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  parBD.Lxy = sim.par.Lxy;
  parBD.H = sim.par.H;
  auto bd = std::make_shared<BD>(sim.pd, parBD);
  bd->addInteractor(poisson);
  bd->forwardTime();
  
  scalar dx = sim.pd->getPos(access::cpu, access::write)[0].x-xp0;
  scalar dy = sim.pd->getPos(access::cpu, access::write)[0].y-yp0;
  scalar dz = sim.pd->getPos(access::cpu, access::write)[0].z-zp0;
  std::cout << "x displacement is " << dx << std::endl;
  std::cout << "y displacement is " << dy << std::endl;
  std::cout << "z displacement is " << dz << std::endl;
  scalar mu_xx = 0.849327755959755;// computed by the DPStokes python code at z = 4R_h above the bottom wall
  scalar mu_yy = 0.849291424072988;
  scalar mu_zz = 0.724165665045591;
  scalar expecteddx = mu_xx*F0x*sim.par.dt;
  scalar expecteddy = mu_yy*F0y*sim.par.dt;
  scalar expecteddz = mu_zz*F0z*sim.par.dt;
  std::cout << "expected x displacement is " << expecteddx << std::endl;
  std::cout << "expected y displacement is " << expecteddy << std::endl;
  std::cout << "expected z displacement is " << expecteddz << std::endl;
  EXPECT_THAT(tolerance(dx, expecteddx), ::testing::IsTrue);
  EXPECT_THAT(tolerance(dy, expecteddy), ::testing::IsTrue);
  EXPECT_THAT(tolerance(dz, expecteddz), ::testing::IsTrue);
}


// Full Wet Mode: Test if an integration works for a pair of particles interacting electrostatically
TEST(FULLWET, Integration){
  UAMMD sim;
  
  sim.par.numberParticles = 2;
  sim.pd = std::make_shared<ParticleData>(sim.par.numberParticles);
  sim.par.Lxy = 76.8;
  sim.par.H = 19.2;
  scalar yp0  = 0.5*sim.par.Lxy;
  scalar yp1  = yp0;
  scalar zp0 = -0.5*sim.par.H+4;
  scalar zp1 = -0.5*sim.par.H+7;
  scalar xp0 = 0.5*sim.par.Lxy+2;
  scalar xp1 = 0.5*sim.par.Lxy-2;
  
  sim.pd->getPos(access::location::cpu, access::mode::write)[0] = {xp0,yp0,zp0};
  sim.pd->getPos(access::location::cpu, access::mode::write)[1] = {xp1,yp1,zp1};
  sim.pd->getCharge(access::location::cpu, access::mode::write)[0] = 1;
  sim.pd->getCharge(access::location::cpu, access::mode::write)[1] = -1;

  real c0 = sim.pd->getCharge(access::location::cpu, access::mode::read)[0];
  real c1 = sim.pd->getCharge(access::location::cpu, access::mode::read)[1];
   
  sim.par.gw = 0.25;
  sim.par.permitivity = 1;
  sim.par.permitivityTop = 0.05;
  sim.par.permitivityBottom = 0.05;
  sim.par.Nxy = 135;
  sim.par.temperature = 0;
  sim.par.viscosity = 1.0/(6*M_PI);
  sim.par.hydrodynamicRadius = 1.0;
  sim.par.wetHydrodynamicRadius = 1.0;
  sim.par.dt = 1;
  sim.par.brownianUpdateRule = "EulerMaruyama";

  using DPP = DPPoissonSlab;
  DPP::Parameters par;
  par.Lxy = make_real2(sim.par.Lxy);
  par.H = sim.par.H;
  par.gw = sim.par.gw;
  DPP::Permitivity perm;
  perm.inside = sim.par.permitivity;
  perm.top = sim.par.permitivityTop;
  perm.bottom = sim.par.permitivityBottom;
  par.permitivity = perm;
  par.Nxy = sim.par.Nxy;
  auto poisson = std::make_shared<DPPoissonSlab>(sim.pd, par);
  thrust::device_vector<real4> field = poisson->computeFieldAtParticles();
  std::vector<real4> fieldAtParticles;
  fieldAtParticles.resize(field.size());
  thrust::copy(field.begin(), field.end(), fieldAtParticles.begin());
  scalar F0x = c0*fieldAtParticles[0].x;
  scalar F0y = c0*fieldAtParticles[0].y;
  scalar F0z = c0*fieldAtParticles[0].z;
  scalar F1x = c1*fieldAtParticles[1].x;
  scalar F1y = c1*fieldAtParticles[1].y;
  scalar F1z = c1*fieldAtParticles[1].z;
  std::cout << "F0x = " << F0x << std::endl;
  std::cout << "F0y = " << F0y << std::endl;
  std::cout << "F0z = " << F0z << std::endl;
  std::cout << "F1x = " << F1x << std::endl;
  std::cout << "F1y = " << F1y << std::endl;
  std::cout << "F1z = " << F1z << std::endl;
  
  using BD = DryWetBD;
  BD::Parameters parBD;
  parBD.temperature = sim.par.temperature;
  parBD.viscosity = sim.par.viscosity;
  parBD.hydrodynamicRadius = sim.par.hydrodynamicRadius;
  parBD.wetRadius = sim.par.wetHydrodynamicRadius;
  parBD.dt = sim.par.dt;
  parBD.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  parBD.Lxy = sim.par.Lxy;
  parBD.H = sim.par.H;
  auto bd = std::make_shared<BD>(sim.pd, parBD);
  bd->addInteractor(poisson);
  bd->forwardTime();

  scalar dx = sim.pd->getPos(access::cpu, access::write)[1].x-xp1;
  scalar dy = sim.pd->getPos(access::cpu, access::write)[1].y-yp1;
  scalar dz = sim.pd->getPos(access::cpu, access::write)[1].z-zp1;
  std::cout << "x displacement is " << dx << std::endl;
  std::cout << "y displacement is " << dy << std::endl;
  std::cout << "z displacement is " << dz << std::endl;
  
  //given by the DPStokes solver code (pair_mobility.py)
  scalar nu_xx = 0.138291767457352;
  scalar nu_xy = 0;
  scalar nu_xz = -0.038706105653531;
  scalar nu_yx = 0;
  scalar nu_yy = 0.047304644896516;
  scalar nu_yz = 0;
  scalar nu_zx = -0.081720265183282;
  scalar nu_zy = 0;
  scalar nu_zz = 0.037554341201021;
  scalar mu_xx = 0.893993039433745;
  scalar mu_yy = 0.893956879655325;
  scalar mu_zz = 0.829029667444467;
  scalar expecteddx = (nu_xx*F0x+nu_xy*F0y+nu_xz*F0z+mu_xx*F1x)*sim.par.dt;
  scalar expecteddy = (nu_yx*F0x+nu_yy*F0y+nu_yz*F0z+mu_yy*F1y)*sim.par.dt;
  scalar expecteddz = (nu_zx*F0x+nu_zy*F0y+nu_zz*F0z+mu_zz*F1z)*sim.par.dt;
  std::cout << "expected x displacement is " << expecteddx << std::endl;
  std::cout << "expected y displacement is " << expecteddy << std::endl;
  std::cout << "expected z displacement is " << expecteddz << std::endl;
  EXPECT_THAT(tolerance(dx, expecteddx), ::testing::IsTrue);
  EXPECT_THAT(tolerance(dy, expecteddy), ::testing::IsTrue);
  EXPECT_THAT(tolerance(dz, expecteddz), ::testing::IsTrue);
}

// ############## Tests by Raul ############## //
TEST(DryWetMobility, CanBeCreated){
  using BD = DryWetBD;
  BD::Parameters par;
  par.temperature = 1.0;
  par.viscosity = 1.0;
  par.hydrodynamicRadius = 1.0;
  par.dt = 1.0;
  par.wetRadius = 0.9;
  par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  par.H = 16;
  par.Lxy = 32;
  auto pd = std::make_shared<ParticleData>(1);
  auto bd = std::make_shared<BD>(pd, par);
}


TEST(FullDryMobility, SelfMobilityIsCorrect){
  using BD = DryWetBD;
  writeDefaultMobilityFile();
  BD::Parameters par;
  par.temperature = 0;
  par.viscosity = 1.0/(6*M_PI);
  par.hydrodynamicRadius = 1.0;
  par.dt = 1.0;
  par.wetRadius = -1;
  par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  par.dryMobilityFile = "uniformMob.dat";
  par.H = 32;
  par.Lxy = 64;
  auto pd = std::make_shared<ParticleData>(1);
  pd->getPos(access::cpu, access::write)[0] = real4();
  auto bd = std::make_shared<BD>(pd, par);
  bd->addInteractor(std::make_shared<miniInteractor>(pd));
  bd->forwardTime();
  real M0 = pd->getPos(access::cpu, access::write)[0].x;
  ASSERT_THAT(M0, ::testing::DoubleNear(1, 1e-5));
}

TEST(FullWetMobility, SelfMobilityIsCorrectAtMiddlePlaneForLargeDomain){
  using BD = DryWetBD;
  writeDefaultMobilityFile();
  BD::Parameters par;
  par.temperature = 0;
  par.viscosity = 1.0/(6*M_PI);
  par.hydrodynamicRadius = 1.0;
  par.dt = 1.0;
  par.wetRadius = par.hydrodynamicRadius; //0<wetRadius<=hydrodynamicRadius means full wet
  par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  par.dryMobilityFile = "uniformMob.dat";
  par.H = 128;
  par.Lxy = 64;
  auto pd = std::make_shared<ParticleData>(1);
  pd->getPos(access::cpu, access::write)[0] = real4();
  auto bd = std::make_shared<BD>(pd, par);
  bd->addInteractor(std::make_shared<miniInteractor>(pd));
  bd->forwardTime();
  real M0 = pd->getPos(access::cpu, access::write)[0].x;
  ASSERT_THAT(M0, ::testing::DoubleNear(1, 1e-1));
}


//Asserts the correctness of the self mobility for a certain wet radius
//All other parameters are hardcoded, see the function.
//The total hydrodynamic radius is 1 (meaning that wetRadius >=1 is full wet)
void computeSelfMobilityWithWetRadius(real wetRadius){
  using BD = DryWetBD;
  writeDefaultMobilityFile();
  BD::Parameters par;
  par.temperature = 0;
  par.viscosity = 1.0/(6*M_PI);
  par.hydrodynamicRadius = 1.0;
  par.dt = 1.0;
  par.wetRadius = wetRadius;
  par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  par.dryMobilityFile = "uniformMob.dat";
  par.H = 64;
  par.Lxy = 64;
  auto pd = std::make_shared<ParticleData>(1);
  pd->getPos(access::cpu, access::write)[0] = real4();
  auto bd = std::make_shared<BD>(pd, par);
  bd->addInteractor(std::make_shared<miniInteractor>(pd));
  bd->forwardTime();
  real M0 = pd->getPos(access::cpu, access::write)[0].x;
  ASSERT_THAT(M0, ::testing::DoubleNear(1, 1e-1))<<"Failed with wet radius "<<wetRadius;
}

TEST(DryWetMobility, SelfMobilityIsCorrectForAnyWetRadius){
  real minWetRadius = 2;
  real maxWetRadius = 16;
  int Ntest = 4;
  fori(0, Ntest){
    real wetRadius = minWetRadius + i*(maxWetRadius - minWetRadius)/(Ntest-1);
    computeSelfMobilityWithWetRadius(wetRadius);
  }
}
