/* Raul P. Pelaez and Aref Hashemi 2023. Tests for the dry wet slit channel Integrator

 */
#include <gtest/gtest.h>
#include "gmock/gmock.h"
#include <random>
#include <fstream>
#include "DryDiffusion.cuh"
#include"Interactor/DoublyPeriodic/DPPoissonSlab.cuh"
using namespace uammd;

struct Parameters{
  int numberParticles;
  real Lxy, H;
  int Nxy = -1;
  int support = 10;
  real numberStandardDeviations = 4;
  real upsampling = 1.2;
  real tolerance = 1e-4;
  real temperature;
  real permitivity, permitivityBottom, permitivityTop;

  real bottomWallSurfaceValue = 0;

  int numberSteps, printSteps, relaxSteps;
  real dt, viscosity, hydrodynamicRadius, wetHydrodynamicRadius;

  real gw;
  real U0, sigma, r_m, p, cutOff;
  real wall_U0, wall_sigma, wall_r_m, wall_p, wall_cutOff;
  real imageDistanceMultiplier;

  std::string outfile, readFile, forcefile, fieldfile;
  std::string mobilityFile;


  std::string brownianUpdateRule = "EulerMaruyama";
  bool idealParticles=false;
  bool noElectrostatics=false;
  int w = 6;
  real beta = 10.13641758;
  int nxy_stokes;
  int nz_stokes;

  real3 externalField;
  int fold;
};


struct UAMMD{
  std::shared_ptr<ParticleData> pd;
  std::shared_ptr<thrust::device_vector<real4>> savedPositions;
  Parameters par;
};

//Adds a force to the first particle, the force defaults to 1,0,0 if not provided
struct miniInteractor: public Interactor{
  real3 f;
public:
  miniInteractor(std::shared_ptr<ParticleData> pd, real3 f = {1,0,0}):
    Interactor(pd), f(f){
  }

  void sum(Computables comp, hipStream_t st =0) override{
    pd->getForce(access::cpu, access::write)[0] = make_real4(f);
  }
};

// ############## Tests by Aref ############## //

// // Add a force to a particle and recompute it using the Intercator
// TEST(Playing, ReadWriteParticleForce){
//   auto pd = std::make_shared<ParticleData>(2);
//   auto external = std::make_shared<miniInteractor>(pd);
//   external->sum({.force=true});
//   real3 F0 = make_real3(pd->getForce(access::location::cpu, access::mode::read)[0]);
//   std::cout << "force on particle #1 = " << F0 << std::endl;
//   EXPECT_THAT(F0.x, ::testing::DoubleNear(1, 1e-5));
// }

// // Reading and writing particle positions
// TEST(Playing, ReadWriteParticlePosition){
//   auto pd = std::make_shared<ParticleData>(2);
//   pd->getPos(access::location::cpu, access::mode::write)[0] = {1,0,0};
//   pd->getPos(access::location::cpu, access::mode::write)[1] = {1.5,0,0};
//   real x0 = pd->getPos(access::location::cpu, access::mode::read)[0].x;
//   real x1 = pd->getPos(access::location::cpu, access::mode::read)[1].x;
//   std::cout << "particle #1 position = " << x0 << std::endl;
//   std::cout << "particle #2 position = " << x1 << std::endl;
//   EXPECT_THAT(x0, ::testing::DoubleNear(1, 1e-5));
//   EXPECT_THAT(x1, ::testing::DoubleNear(1.5, 1e-5));
// }

// // Compute electrostatic fields for 2 particles
// TEST(FULLDRY, ComputeElectrostaticField){
//   using DPP = DPPoissonSlab;
//   DPP::Parameters par;
//   // needs a real2 for Lxy (see DPPoissonSlab.cuh)
//   par.Lxy = {32, 32};
//   par.H = 20;
//   par.gw = 0.25;
//   DPP::Permitivity perm;
//   perm.inside = 1.0;
//   perm.top = 1.0;
//   perm.bottom = 0.05;
//   par.permitivity = perm;
//   par.Nxy = 72;
  
//   auto pd = std::make_shared<ParticleData>(2);
//   pd->getPos(access::location::cpu, access::mode::write)[0] = {6.001,4.02,0.05};
//   pd->getPos(access::location::cpu, access::mode::write)[1] = {5.11,5.24,-2.1};
//   pd->getCharge(access::location::cpu, access::mode::write)[0] = 1;
//   pd->getCharge(access::location::cpu, access::mode::write)[1] = -1;

//   real c0 = pd->getCharge(access::location::cpu, access::mode::read)[0];
//   std::cout << "charge of particle #1 = " << c0 << std::endl;
//   real c1 = pd->getCharge(access::location::cpu, access::mode::read)[1];
//   std::cout << "charge of particle #2 = " << c1 << std::endl;
  
//   auto poisson = std::make_shared<DPPoissonSlab>(pd, par);
//   thrust::device_vector<real4> field = poisson->computeFieldAtParticles();
//   std::vector<real4> fieldAtParticles;
//   fieldAtParticles.resize(field.size());
//   thrust::copy(field.begin(), field.end(), fieldAtParticles.begin());
//   real E0x = fieldAtParticles[0].x;
//   std::cout << "x field at particle #1 (electrostatics) = " << E0x << std::endl;
//   real magicalValue = -0.003902211008208;//given by the MATLB code
//   EXPECT_THAT(E0x, ::testing::DoubleNear(magicalValue, 1e-5));
// }

// // Test if a simple integration is working properly
// TEST(FULLDRY, SimpleIntegration){
//   using BD = DryWetBD;
//   BD::Parameters par;
//   par.temperature = 0;
//   // \mu = 1/(6\pi\eta a) = 1
//   par.viscosity = 1.0/(6*M_PI);
//   par.hydrodynamicRadius = 1.0;
//   par.dt = 1.0;
//   par.wetRadius = -1;
//   par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
//   par.H = 32;
//   par.Lxy = 64;
//   auto pd = std::make_shared<ParticleData>(1);
//   pd->getPos(access::cpu, access::write)[0] = {0,0,0};
//   auto bd = std::make_shared<BD>(pd, par);
//   bd->addInteractor(std::make_shared<miniInteractor>(pd));
//   bd->forwardTime();
//   // F = [1, 0, 0]; \mu = 1; U = 1; \Delta t = 1. \Rightarrow \Delta x = 1
//   real dx = pd->getPos(access::cpu, access::write)[0].x;
//   EXPECT_THAT(dx, ::testing::DoubleNear(1, 1e-5));
// }

// Test if an integration works for a pair of particles interacting electrostatically
TEST(FULLDRY, Integration){
  UAMMD sim;
  
  sim.par.numberParticles = 2;
  sim.pd = std::make_shared<ParticleData>(sim.par.numberParticles);
  
  sim.pd->getPos(access::location::cpu, access::mode::write)[0] = {6.001,4.02,0.05};
  sim.pd->getPos(access::location::cpu, access::mode::write)[1] = {5.11,5.24,-2.1};
  sim.pd->getCharge(access::location::cpu, access::mode::write)[0] = 1;
  sim.pd->getCharge(access::location::cpu, access::mode::write)[1] = -1;

  sim.par.Lxy = 32;
  sim.par.H = 20;
  sim.par.gw = 0.25;
  sim.par.permitivity = 1;
  sim.par.permitivityTop = 1;
  sim.par.permitivityBottom = 0.05;
  sim.par.Nxy = 72;
  sim.par.temperature = 0;
  sim.par.viscosity = 1.0/(6*M_PI);
  sim.par.hydrodynamicRadius = 1.0;
  sim.par.wetHydrodynamicRadius = -1;
  sim.par.dt = 0.01;
  sim.par.brownianUpdateRule = "EulerMaruyama";

  using DPP = DPPoissonSlab;
  DPP::Parameters par;
  par.Lxy = make_real2(sim.par.Lxy);
  par.H = sim.par.H;
  par.gw = sim.par.gw;
  DPP::Permitivity perm;
  perm.inside = sim.par.permitivity;
  perm.top = sim.par.permitivityTop;
  perm.bottom = sim.par.permitivityBottom;
  par.permitivity = perm;
  par.Nxy = sim.par.Nxy;
  auto poisson = std::make_shared<DPPoissonSlab>(sim.pd, par);
  
  using BD = DryWetBD;
  BD::Parameters parBD;
  parBD.temperature = sim.par.temperature;
  parBD.viscosity = sim.par.viscosity;
  parBD.hydrodynamicRadius = sim.par.hydrodynamicRadius;
  parBD.wetRadius = sim.par.wetHydrodynamicRadius;
  parBD.dt = sim.par.dt;
  parBD.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  parBD.Lxy = sim.par.Lxy;
  parBD.H = sim.par.H;
  auto bd = std::make_shared<BD>(sim.pd, parBD);
  bd->addInteractor(poisson);
  bd->forwardTime();

  real dx = sim.pd->getPos(access::cpu, access::write)[0].x-6.001;
  std::cout << "displacement is " << dx << std::endl;
  real magicalValue = -0.0000390221100821;//given by the MATLB code
  EXPECT_THAT(dx, ::testing::DoubleNear(magicalValue, 1e-5));
}


// // ############## Tests by Raul ############## //
// TEST(DryWetMobility, CanBeCreated){
//   using BD = DryWetBD;
//   BD::Parameters par;
//   par.temperature = 1.0;
//   par.viscosity = 1.0;
//   par.hydrodynamicRadius = 1.0;
//   par.dt = 1.0;
//   par.wetRadius = 0.9;
//   par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
//   //par.dryMobilityFile = sim.par.mobilityFile;
//   par.H = 16;
//   par.Lxy = 32;
//   auto pd = std::make_shared<ParticleData>(1);
//   auto bd = std::make_shared<BD>(pd, par);
// }


// TEST(FullDryMobility, SelfMobilityIsCorrect){
//   using BD = DryWetBD;
//   writeDefaultMobilityFile();
//   BD::Parameters par;
//   par.temperature = 0;
//   par.viscosity = 1.0/(6*M_PI);
//   par.hydrodynamicRadius = 1.0;
//   par.dt = 1.0;
//   par.wetRadius = -1;
//   par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
//   par.dryMobilityFile = "mob.dat";
//   par.H = 32;
//   par.Lxy = 64;
//   auto pd = std::make_shared<ParticleData>(1);
//   pd->getPos(access::cpu, access::write)[0] = real4();
//   auto bd = std::make_shared<BD>(pd, par);
//   bd->addInteractor(std::make_shared<miniInteractor>(pd));
//   bd->forwardTime();
//   real M0 = pd->getPos(access::cpu, access::write)[0].x;
//   ASSERT_THAT(M0, ::testing::DoubleNear(1, 1e-5));
// }

// TEST(FullWetMobility, SelfMobilityIsCorrectAtMiddlePlaneForLargeDomain){
//   using BD = DryWetBD;
//   writeDefaultMobilityFile();
//   BD::Parameters par;
//   par.temperature = 0;
//   par.viscosity = 1.0/(6*M_PI);
//   par.hydrodynamicRadius = 1.0;
//   par.dt = 1.0;
//   par.wetRadius = par.hydrodynamicRadius; //0<wetRadius<=hydrodynamicRadius means full wet
//   par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
//   par.dryMobilityFile = "mob.dat";
//   par.H = 128;
//   par.Lxy = 64;
//   auto pd = std::make_shared<ParticleData>(1);
//   pd->getPos(access::cpu, access::write)[0] = real4();
//   auto bd = std::make_shared<BD>(pd, par);
//   bd->addInteractor(std::make_shared<miniInteractor>(pd));
//   bd->forwardTime();
//   real M0 = pd->getPos(access::cpu, access::write)[0].x;
//   ASSERT_THAT(M0, ::testing::DoubleNear(1, 1e-1));
// }


// //Asserts the correctness of the self mobility for a certain wet radius
// //All other parameters are hardcoded, see the function.
// //The total hydrodynamic radius is 1 (meaning that wetRadius >=1 is full wet)
// void computeSelfMobilityWithWetRadius(real wetRadius){
//   using BD = DryWetBD;
//   writeDefaultMobilityFile();
//   BD::Parameters par;
//   par.temperature = 0;
//   par.viscosity = 1.0/(6*M_PI);
//   par.hydrodynamicRadius = 1.0;
//   par.dt = 1.0;
//   par.wetRadius = wetRadius;
//   par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
//   par.dryMobilityFile = "mob.dat";
//   par.H = 64;
//   par.Lxy = 64;
//   auto pd = std::make_shared<ParticleData>(1);
//   pd->getPos(access::cpu, access::write)[0] = real4();
//   auto bd = std::make_shared<BD>(pd, par);
//   bd->addInteractor(std::make_shared<miniInteractor>(pd));
//   bd->forwardTime();
//   real M0 = pd->getPos(access::cpu, access::write)[0].x;
//   ASSERT_THAT(M0, ::testing::DoubleNear(1, 1e-1))<<"Failed with wet radius "<<wetRadius;
// }

// TEST(DryWetMobility, SelfMobilityIsCorrectForAnyWetRadius){
//   real minWetRadius = 2;
//   real maxWetRadius = 16;
//   int Ntest = 4;
//   fori(0, Ntest){
//     real wetRadius = minWetRadius + i*(maxWetRadius - minWetRadius)/(Ntest-1);
//     computeSelfMobilityWithWetRadius(wetRadius);
//   }
// }
