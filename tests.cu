#include <gtest/gtest.h>
#include "gmock/gmock.h"
#include <random>
#include <fstream>
#include "DryDiffusion.cuh"
using namespace uammd;

//Writes a mobility file with constant mobility accross the domain
void writeDefaultMobilityFile(){
  std::ofstream out("mob.dat");
  out<<"-1.0 1.0 1.0 1.0"<<std::endl;
  out<<"0.0 1.0 1.0 1.0"<<std::endl;
  out<<"1.0 1.0 1.0 1.0"<<std::endl;
}

TEST(DryWetMobility, CanBeCreated){
  using BD = DryWetBD;
  BD::Parameters par;
  par.temperature = 1.0;
  par.viscosity = 1.0;
  par.hydrodynamicRadius = 1.0;
  par.dt = 1.0;
  par.wetRadius = 0.9;
  par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  //par.dryMobilityFile = sim.par.mobilityFile;
  par.H = 16;
  par.Lxy = 32;
  auto pd = std::make_shared<ParticleData>(1);
  auto bd = std::make_shared<BD>(pd, par);
}

//Adds a force to the first particle, the force defaults to 1,0,0 of not provided
struct miniInteractor: public Interactor{
  real3 f;
public:
  miniInteractor(std::shared_ptr<ParticleData> pd, real3 f = {1,0,0}):
    Interactor(pd), f(f){
  }

  void sum(Computables comp, hipStream_t st =0) override{
    pd->getForce(access::cpu, access::write)[0] = make_real4(f);
  }
};


TEST(FullDryMobility, SelfMobilityIsCorrect){
  using BD = DryWetBD;
  writeDefaultMobilityFile();
  BD::Parameters par;
  par.temperature = 0;
  par.viscosity = 1.0/(6*M_PI);
  par.hydrodynamicRadius = 1.0;
  par.dt = 1.0;
  par.wetRadius = -1;
  par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  par.dryMobilityFile = "mob.dat";
  par.H = 32;
  par.Lxy = 64;
  auto pd = std::make_shared<ParticleData>(1);
  pd->getPos(access::cpu, access::write)[0] = real4();
  auto bd = std::make_shared<BD>(pd, par);
  bd->addInteractor(std::make_shared<miniInteractor>(pd));
  bd->forwardTime();
  real M0 = pd->getPos(access::cpu, access::write)[0].x;
  ASSERT_THAT(M0, ::testing::DoubleNear(1, 1e-5));
}



TEST(FullWetMobility, SelfMobilityIsCorrectAtMiddlePlaneForLargeDomain){
  using BD = DryWetBD;
  writeDefaultMobilityFile();
  BD::Parameters par;
  par.temperature = 0;
  par.viscosity = 1.0/(6*M_PI);
  par.hydrodynamicRadius = 1.0;
  par.dt = 1.0;
  par.wetRadius = par.hydrodynamicRadius;
  par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  par.dryMobilityFile = "mob.dat";
  par.H = 128;
  par.Lxy = 64;
  auto pd = std::make_shared<ParticleData>(1);
  pd->getPos(access::cpu, access::write)[0] = real4();
  auto bd = std::make_shared<BD>(pd, par);
  bd->addInteractor(std::make_shared<miniInteractor>(pd));
  bd->forwardTime();
  real M0 = pd->getPos(access::cpu, access::write)[0].x;
  ASSERT_THAT(M0, ::testing::DoubleNear(1, 1e-2));
}
