#include "hip/hip_runtime.h"
/* Raul P. Pelaez and Aref Hashemi 2023. Tests for the dry wet slit channel Integrator

 */
#include <gtest/gtest.h>
#include "gmock/gmock.h"
#include <random>
#include <fstream>
#include "DryDiffusion.cuh"
#include"Interactor/DoublyPeriodic/DPPoissonSlab.cuh"
using namespace uammd;

using scalar = double;

// A normalized measure for accuracy
scalar tolerance(scalar correctVal, scalar mult){
  scalar measure;
  if (abs(correctVal) > 1e-15){
    measure = mult*abs(correctVal);
  } else {
    measure = 1e-15;
  }

  return measure;
}

//Writes a mobility file with constant mobility accross the domain
void writeDefaultMobilityFile(){
  std::ofstream out("uniformMob.dat");
  out<<"-1.0 1.0 1.0 1.0"<<std::endl;
  out<<"0.0 1.0 1.0 1.0"<<std::endl;
  out<<"1.0 1.0 1.0 1.0"<<std::endl;
}

struct Parameters{
  int numberParticles;
  real Lxy, H;
  int Nxy = -1;
  int support = 10;
  real numberStandardDeviations = 4;
  real upsampling = 1.2;
  real tolerance = 1e-4;
  real temperature;
  real permitivity, permitivityBottom, permitivityTop;

  real bottomWallSurfaceValue = 0;

  int numberSteps, printSteps, relaxSteps;
  real dt, viscosity, hydrodynamicRadius, wetHydrodynamicRadius;

  real gw;
  real U0, sigma, r_m, p, cutOff;
  real wall_U0, wall_sigma, wall_r_m, wall_p, wall_cutOff;
  real imageDistanceMultiplier;

  std::string outfile, readFile, forcefile, fieldfile;
  std::string mobilityFile;


  std::string brownianUpdateRule = "EulerMaruyama";
  bool idealParticles=false;
  bool noElectrostatics=false;
  int w = 6;
  real beta = 10.13641758;
  int nxy_stokes;
  int nz_stokes;

  real3 externalField;
  int fold;
};


struct UAMMD{
  std::shared_ptr<ParticleData> pd;
  std::shared_ptr<thrust::device_vector<real4>> savedPositions;
  Parameters par;
};

//Adds a force to the first particle, the force defaults to 1,0,0 if not provided
struct miniInteractor: public Interactor{
  real3 f;
public:
  miniInteractor(std::shared_ptr<ParticleData> pd, real3 f = {1,0,0}):
    Interactor(pd), f(f){
  }

  void sum(Computables comp, hipStream_t st =0) override{
    pd->getForce(access::cpu, access::write)[0] = make_real4(f);
  }
};

// ############## Tests by Aref ############## //

// Add a force to a particle and recompute it using the Intercator
TEST(Playing, ReadWriteParticleForce){
  auto pd = std::make_shared<ParticleData>(2);
  auto external = std::make_shared<miniInteractor>(pd);
  external->sum({.force=true});
  real3 F0 = make_real3(pd->getForce(access::location::cpu, access::mode::read)[0]);
  std::cout << "force on particle #1 = " << F0 << std::endl;
  EXPECT_THAT(F0.x, ::testing::DoubleNear(1, 1e-5));
}

// Reading and writing particle positions
TEST(Playing, ReadWriteParticlePosition){
  auto pd = std::make_shared<ParticleData>(2);
  pd->getPos(access::location::cpu, access::mode::write)[0] = {1,0,0};
  pd->getPos(access::location::cpu, access::mode::write)[1] = {1.5,0,0};
  real x0 = pd->getPos(access::location::cpu, access::mode::read)[0].x;
  real x1 = pd->getPos(access::location::cpu, access::mode::read)[1].x;
  std::cout << "particle #1 position = " << x0 << std::endl;
  std::cout << "particle #2 position = " << x1 << std::endl;
  EXPECT_THAT(x0, ::testing::DoubleNear(1, 1e-5));
  EXPECT_THAT(x1, ::testing::DoubleNear(1.5, 1e-5));
}

// full dry mode: Compute electrostatic fields for 2 particles
TEST(FULLDRY, ComputeElectrostaticField){
  using DPP = DPPoissonSlab;
  DPP::Parameters par;
  // needs a real2 for Lxy (see DPPoissonSlab.cuh)
  par.Lxy = {76.8, 76.8};
  par.H = 19.2;
  par.gw = 0.25;
  DPP::Permitivity perm;
  perm.inside = 1.0;
  perm.top = 1.0;
  perm.bottom = 0.05;
  par.permitivity = perm;
  par.Nxy = 72;
  
  auto pd = std::make_shared<ParticleData>(2);
  pd->getPos(access::location::cpu, access::mode::write)[0] = {7,6,-4.8};
  pd->getPos(access::location::cpu, access::mode::write)[1] = {11,6,-4.8};
  pd->getCharge(access::location::cpu, access::mode::write)[0] = 1;
  pd->getCharge(access::location::cpu, access::mode::write)[1] = -1;

  real c0 = pd->getCharge(access::location::cpu, access::mode::read)[0];
  std::cout << "charge of particle #1 = " << c0 << std::endl;
  real c1 = pd->getCharge(access::location::cpu, access::mode::read)[1];
  std::cout << "charge of particle #2 = " << c1 << std::endl;
  
  auto poisson = std::make_shared<DPPoissonSlab>(pd, par);
  thrust::device_vector<real4> field = poisson->computeFieldAtParticles();
  std::vector<real4> fieldAtParticles;
  fieldAtParticles.resize(field.size());
  thrust::copy(field.begin(), field.end(), fieldAtParticles.begin());
  real E0x = fieldAtParticles[0].x;
  real E0y = fieldAtParticles[0].y;
  real E0z = fieldAtParticles[0].z;
  std::cout << "x field at particle #1 (electrostatics) = " << E0x << std::endl;
  std::cout << "y field at particle #1 (electrostatics) = " << E0y << std::endl;
  std::cout << "z field at particle #1 (electrostatics) = " << E0z << std::endl;
  real magicalValuex = 0.005223629199127;//given by the MATLB code
  real magicalValuey = 0                ;
  real magicalValuez = 0.000166699457402;
  EXPECT_THAT(E0x, ::testing::DoubleNear(magicalValuex, tolerance(magicalValuex,1e-5)));
  EXPECT_THAT(E0y, ::testing::DoubleNear(magicalValuey, tolerance(magicalValuey,1e-5)));
  EXPECT_THAT(E0z, ::testing::DoubleNear(magicalValuez, tolerance(magicalValuez,1e-5)));
}

// full dry mode: Test if a simple integration is working properly
TEST(FULLDRY, SimpleIntegration){
  using BD = DryWetBD;
  BD::Parameters par;
  writeDefaultMobilityFile();
  par.dryMobilityFile = "uniformMob.dat";
  par.temperature = 0;
  // \mu = 1/(6\pi\eta a) = 1
  par.viscosity = 1.0/(6*M_PI);
  par.hydrodynamicRadius = 1.0;
  par.dt = 1.0;
  par.wetRadius = -1;
  par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  par.H = 19.2;
  par.Lxy = 76.8;
  auto pd = std::make_shared<ParticleData>(1);
  pd->getPos(access::cpu, access::write)[0] = {0,0,0};
  auto bd = std::make_shared<BD>(pd, par);
  bd->addInteractor(std::make_shared<miniInteractor>(pd));
  bd->forwardTime();
  // F = [1, 0, 0]; \mu = 1; U = 1; \Delta t = 1. \Rightarrow \Delta x = 1
  real dx = pd->getPos(access::cpu, access::write)[0].x;
  EXPECT_THAT(dx, ::testing::DoubleNear(1, 1e-5));
}

// full dry mode: Test if an integration works for a pair of particles interacting electrostatically
// self mobility given by Einstein equation
TEST(FULLDRY, IntegrationFlatMobility){
  UAMMD sim;
  
  sim.par.numberParticles = 2;
  sim.pd = std::make_shared<ParticleData>(sim.par.numberParticles);
  
  sim.pd->getPos(access::location::cpu, access::mode::write)[0] = {7,6,-4.8};
  sim.pd->getPos(access::location::cpu, access::mode::write)[1] = {11,6,-4.8};
  sim.pd->getCharge(access::location::cpu, access::mode::write)[0] = 1;
  sim.pd->getCharge(access::location::cpu, access::mode::write)[1] = -1;

  // {
  //   auto pos = sim.pd->getPos(access::cpu, access::write);
  //   pos[0] = {1,2,3};
  //   pos[1] = {2,3,4};
  // }
  
  sim.par.Lxy = 76.8;
  sim.par.H = 19.2;
  sim.par.gw = 0.25;
  sim.par.permitivity = 1;
  sim.par.permitivityTop = 1;
  sim.par.permitivityBottom = 0.05;
  sim.par.Nxy = 72;
  sim.par.temperature = 0;
  sim.par.viscosity = 1.0/(6*M_PI);
  sim.par.hydrodynamicRadius = 1.0;
  sim.par.wetHydrodynamicRadius = -1;
  sim.par.dt = 0.01;
  sim.par.brownianUpdateRule = "EulerMaruyama";

  using DPP = DPPoissonSlab;
  DPP::Parameters par;
  par.Lxy = make_real2(sim.par.Lxy);
  par.H = sim.par.H;
  par.gw = sim.par.gw;
  DPP::Permitivity perm;
  perm.inside = sim.par.permitivity;
  perm.top = sim.par.permitivityTop;
  perm.bottom = sim.par.permitivityBottom;
  par.permitivity = perm;
  par.Nxy = sim.par.Nxy;
  auto poisson = std::make_shared<DPPoissonSlab>(sim.pd, par);
  thrust::device_vector<real4> field = poisson->computeFieldAtParticles();
  std::vector<real4> fieldAtParticles;
  fieldAtParticles.resize(field.size());
  thrust::copy(field.begin(), field.end(), fieldAtParticles.begin());
  scalar F0x = fieldAtParticles[0].x;
  scalar F0y = fieldAtParticles[0].y;
  scalar F0z = fieldAtParticles[0].z;
  
  using BD = DryWetBD;
  BD::Parameters parBD;
  writeDefaultMobilityFile();
  parBD.dryMobilityFile = "uniformMob.dat";
  parBD.temperature = sim.par.temperature;
  parBD.viscosity = sim.par.viscosity;
  parBD.hydrodynamicRadius = sim.par.hydrodynamicRadius;
  parBD.wetRadius = sim.par.wetHydrodynamicRadius;
  parBD.dt = sim.par.dt;
  parBD.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  parBD.Lxy = sim.par.Lxy;
  parBD.H = sim.par.H;
  auto bd = std::make_shared<BD>(sim.pd, parBD);
  bd->addInteractor(poisson);
  bd->forwardTime();

  scalar dx = sim.pd->getPos(access::cpu, access::write)[0].x-7;
  scalar dy = sim.pd->getPos(access::cpu, access::write)[0].y-6;
  scalar dz = sim.pd->getPos(access::cpu, access::write)[0].z+4.8;
  std::cout << "x displacement is " << dx << std::endl;
  std::cout << "y displacement is " << dy << std::endl;
  std::cout << "z displacement is " << dz << std::endl;
  scalar magicalValuedx = F0x*1*sim.par.dt;//0.0000522362919913 given by the MATLB code  
  scalar magicalValuedy = F0y*1*sim.par.dt;//0
  scalar magicalValuedz = F0z*1*sim.par.dt;//0.0000016669945740;
  EXPECT_THAT(dx, ::testing::DoubleNear(magicalValuedx, tolerance(magicalValuedx,1e-5)));
  EXPECT_THAT(dy, ::testing::DoubleNear(magicalValuedy, tolerance(magicalValuedy,1e-5)));
  EXPECT_THAT(dz, ::testing::DoubleNear(magicalValuedz, tolerance(magicalValuedz,1e-5)));
}


// test is the self mobility is computed correctly
TEST(FULLDRY, selfMobility){
  auto pd = std::make_shared<ParticleData>(1);//dummy instance
  using BD = DryWetBD;
  BD::Parameters parBD;
  parBD.viscosity = 1.0/(6*M_PI);
  parBD.hydrodynamicRadius = 1;
  parBD.Lxy = 76.8;
  parBD.H = 19.2;
  auto bd = std::make_shared<BD>(pd, parBD);

  real z = 4;
  real4 mobilityData = computeSelfMobility(parBD, z);
  scalar muxx = mobilityData.y;
  scalar muyy = mobilityData.z;
  scalar muzz = mobilityData.w;
  std::cout << muxx << std::endl;
  std::cout << muyy << std::endl;
  std::cout << muzz << std::endl;
  scalar magicalValuemuxx = 0.862495177070900;// computed by the DPStokes python code at z = 4R_h above the bottom wall
  scalar magicalValuemuyy = 0.862458866786545;
  scalar magicalValuemuzz = 0.725839526229897;
  std::cout << "expected xx mobility is " << magicalValuemuxx << std::endl;
  std::cout << "expected yy mobility is " << magicalValuemuyy << std::endl;
  std::cout << "expected zz mobility is " << magicalValuemuzz << std::endl;
  EXPECT_THAT(muxx, ::testing::DoubleNear(magicalValuemuxx, tolerance(magicalValuemuxx,1e-2)));
  EXPECT_THAT(muyy, ::testing::DoubleNear(magicalValuemuyy, tolerance(magicalValuemuyy,1e-2)));
  EXPECT_THAT(muzz, ::testing::DoubleNear(magicalValuemuzz, tolerance(magicalValuemuzz,1e-2)));
}


// full dry mode: Test if an integration works for a pair of particles interacting electrostatically
// self mobility precomputed by the Stokes solver
TEST(FULLDRY, Integration){
  UAMMD sim;
  
  sim.par.numberParticles = 2;
  sim.pd = std::make_shared<ParticleData>(sim.par.numberParticles);
  
  sim.par.Lxy = 76.8;
  sim.par.H = 19.2;
  scalar yp0  = 0.5*sim.par.Lxy;
  scalar yp1  = yp0;
  scalar zp0 = -0.5*sim.par.H+4;
  scalar zp1 = -0.5*sim.par.H+7;
  scalar xp0 = 0.5*sim.par.Lxy+2;
  scalar xp1 = 0.5*sim.par.Lxy-2;
  
  sim.pd->getPos(access::location::cpu, access::mode::write)[0] = {xp0,yp0,zp0};
  sim.pd->getPos(access::location::cpu, access::mode::write)[1] = {xp1,yp1,zp1};
  sim.pd->getCharge(access::location::cpu, access::mode::write)[0] = 1;
  sim.pd->getCharge(access::location::cpu, access::mode::write)[1] = -1;

  sim.par.gw = 0.25;
  sim.par.permitivity = 1;
  sim.par.permitivityTop = 1;
  sim.par.permitivityBottom = 0.05;
  sim.par.Nxy = 72;
  sim.par.temperature = 0;
  sim.par.viscosity = 1.0/(6*M_PI);
  sim.par.hydrodynamicRadius = 1.0;
  sim.par.wetHydrodynamicRadius = -1;
  sim.par.dt = 0.01;
  sim.par.brownianUpdateRule = "EulerMaruyama";

  using DPP = DPPoissonSlab;
  DPP::Parameters par;
  par.Lxy = make_real2(sim.par.Lxy);
  par.H = sim.par.H;
  par.gw = sim.par.gw;
  DPP::Permitivity perm;
  perm.inside = sim.par.permitivity;
  perm.top = sim.par.permitivityTop;
  perm.bottom = sim.par.permitivityBottom;
  par.permitivity = perm;
  par.Nxy = sim.par.Nxy;
  auto poisson = std::make_shared<DPPoissonSlab>(sim.pd, par);
  thrust::device_vector<real4> field = poisson->computeFieldAtParticles();
  std::vector<real4> fieldAtParticles;
  fieldAtParticles.resize(field.size());
  thrust::copy(field.begin(), field.end(), fieldAtParticles.begin());
  scalar F0x = fieldAtParticles[0].x;
  scalar F0y = fieldAtParticles[0].y;
  scalar F0z = fieldAtParticles[0].z;

  
  using BD = DryWetBD;
  BD::Parameters parBD;
  // writeDefaultMobilityFile();
  // parBD.dryMobilityFile = "uniformMob.dat";
  parBD.temperature = sim.par.temperature;
  parBD.viscosity = sim.par.viscosity;
  parBD.hydrodynamicRadius = sim.par.hydrodynamicRadius;
  parBD.wetRadius = sim.par.wetHydrodynamicRadius;
  parBD.dt = sim.par.dt;
  parBD.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  parBD.Lxy = sim.par.Lxy;
  parBD.H = sim.par.H;
  auto bd = std::make_shared<BD>(sim.pd, parBD);
  bd->addInteractor(poisson);
  bd->forwardTime();
  
  scalar dx = sim.pd->getPos(access::cpu, access::write)[0].x-xp0;
  scalar dy = sim.pd->getPos(access::cpu, access::write)[0].y-yp0;
  scalar dz = sim.pd->getPos(access::cpu, access::write)[0].z-zp0;
  std::cout << "x displacement is " << dx << std::endl;
  std::cout << "y displacement is " << dy << std::endl;
  std::cout << "z displacement is " << dz << std::endl;
  scalar mu_xx = 0.862495177070900;//Computed by DPStokes solver
  scalar mu_yy = 0.862458866786545;
  scalar mu_zz = 0.725839526229897;
  scalar magicalValuedx = mu_xx*F0x*sim.par.dt;
  scalar magicalValuedy = mu_yy*F0y*sim.par.dt;
  scalar magicalValuedz = mu_zz*F0z*sim.par.dt;
  std::cout << "expected x displacement is " << magicalValuedx << std::endl;
  std::cout << "expected y displacement is " << magicalValuedy << std::endl;
  std::cout << "expected z displacement is " << magicalValuedz << std::endl;
  EXPECT_THAT(dx, ::testing::DoubleNear(magicalValuedx, tolerance(magicalValuedx,1e-2)));
  EXPECT_THAT(dy, ::testing::DoubleNear(magicalValuedy, tolerance(magicalValuedy,1e-2)));
  EXPECT_THAT(dz, ::testing::DoubleNear(magicalValuedz, tolerance(magicalValuedz,1e-2)));
}


// full wet mode: Test if an integration works for a pair of particles interacting electrostatically
TEST(FULLWET, Integration){
  UAMMD sim;
  
  sim.par.numberParticles = 2;
  sim.pd = std::make_shared<ParticleData>(sim.par.numberParticles);
  sim.par.Lxy = 76.8;
  sim.par.H = 19.2;
  scalar yp0  = 0.5*sim.par.Lxy;
  scalar yp1  = yp0;
  scalar zp0 = -0.5*sim.par.H+4;
  scalar zp1 = -0.5*sim.par.H+7;
  scalar xp0 = 0.5*sim.par.Lxy+2;
  scalar xp1 = 0.5*sim.par.Lxy-2;
  
  sim.pd->getPos(access::location::cpu, access::mode::write)[0] = {xp0,yp0,zp0};
  sim.pd->getPos(access::location::cpu, access::mode::write)[1] = {xp1,yp1,zp1};
  sim.pd->getCharge(access::location::cpu, access::mode::write)[0] = 1;
  sim.pd->getCharge(access::location::cpu, access::mode::write)[1] = -1;

  real c0 = sim.pd->getCharge(access::location::cpu, access::mode::read)[0];
  real c1 = sim.pd->getCharge(access::location::cpu, access::mode::read)[1];
   
  sim.par.gw = 0.25;
  sim.par.permitivity = 1;
  sim.par.permitivityTop = 1;
  sim.par.permitivityBottom = 0.05;
  sim.par.Nxy = 135;
  sim.par.temperature = 0;
  sim.par.viscosity = 1.0/(6*M_PI);
  sim.par.hydrodynamicRadius = 1.0;
  sim.par.wetHydrodynamicRadius = 1.0;
  sim.par.dt = 1;
  sim.par.brownianUpdateRule = "EulerMaruyama";

  using DPP = DPPoissonSlab;
  DPP::Parameters par;
  par.Lxy = make_real2(sim.par.Lxy);
  par.H = sim.par.H;
  par.gw = sim.par.gw;
  DPP::Permitivity perm;
  perm.inside = sim.par.permitivity;
  perm.top = sim.par.permitivityTop;
  perm.bottom = sim.par.permitivityBottom;
  par.permitivity = perm;
  par.Nxy = sim.par.Nxy;
  auto poisson = std::make_shared<DPPoissonSlab>(sim.pd, par);
  thrust::device_vector<real4> field = poisson->computeFieldAtParticles();
  std::vector<real4> fieldAtParticles;
  fieldAtParticles.resize(field.size());
  thrust::copy(field.begin(), field.end(), fieldAtParticles.begin());
  scalar F0x = c0*fieldAtParticles[0].x;
  scalar F0y = c0*fieldAtParticles[0].y;
  scalar F0z = c0*fieldAtParticles[0].z;
  scalar F1x = c1*fieldAtParticles[1].x;
  scalar F1y = c1*fieldAtParticles[1].y;
  scalar F1z = c1*fieldAtParticles[1].z;
  std::cout << "F0x = " << F0x << std::endl;
  std::cout << "F0y = " << F0y << std::endl;
  std::cout << "F0z = " << F0z << std::endl;
  std::cout << "F1x = " << F1x << std::endl;
  std::cout << "F1y = " << F1y << std::endl;
  std::cout << "F1z = " << F1z << std::endl;
  
  using BD = DryWetBD;
  BD::Parameters parBD;
  parBD.temperature = sim.par.temperature;
  parBD.viscosity = sim.par.viscosity;
  parBD.hydrodynamicRadius = sim.par.hydrodynamicRadius;
  parBD.wetRadius = sim.par.wetHydrodynamicRadius;
  parBD.dt = sim.par.dt;
  parBD.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
  parBD.Lxy = sim.par.Lxy;
  parBD.H = sim.par.H;
  auto bd = std::make_shared<BD>(sim.pd, parBD);
  bd->addInteractor(poisson);
  bd->forwardTime();

  scalar dx = sim.pd->getPos(access::cpu, access::write)[1].x-xp1;
  scalar dy = sim.pd->getPos(access::cpu, access::write)[1].y-yp1;
  scalar dz = sim.pd->getPos(access::cpu, access::write)[1].z-zp1;
  std::cout << "x displacement is " << dx << std::endl;
  std::cout << "y displacement is " << dy << std::endl;
  std::cout << "z displacement is " << dz << std::endl;
  
  //given by the DPStokes solver code (pair_mobility.py)
  scalar nu_xx = 0.156615987796052 ;
  scalar nu_xy = 0                 ;
  scalar nu_xz = -0.036746481747739;
  scalar nu_yx = 0                 ;
  scalar nu_yy = 0.066428530938183 ;
  scalar nu_yz = 0                 ;
  scalar nu_zx = -0.085289117508094;
  scalar nu_zy = 0                 ;
  scalar nu_zz = 0.041363902842424 ;
  scalar mu_xx = 0.924166257588772 ;
  scalar mu_yy = 0.924130139892737 ;
  scalar mu_zz = 0.840468139888785 ;
  scalar magicalValuedx = (nu_xx*F0x+nu_xy*F0y+nu_xz*F0z+mu_xx*F1x)*sim.par.dt;
  scalar magicalValuedy = (nu_yx*F0x+nu_yy*F0y+nu_yz*F0z+mu_yy*F1y)*sim.par.dt;
  scalar magicalValuedz = (nu_zx*F0x+nu_zy*F0y+nu_zz*F0z+mu_zz*F1z)*sim.par.dt;
  std::cout << "expected x displacement is " << magicalValuedx << std::endl;
  std::cout << "expected y displacement is " << magicalValuedy << std::endl;
  std::cout << "expected z displacement is " << magicalValuedz << std::endl;
  
  EXPECT_THAT(dx, ::testing::DoubleNear(magicalValuedx, tolerance(magicalValuedx,1e-2)));
  EXPECT_THAT(dy, ::testing::DoubleNear(magicalValuedy, tolerance(magicalValuedy,1e-2)));
  EXPECT_THAT(dz, ::testing::DoubleNear(magicalValuedz, tolerance(magicalValuedz,1e-2)));
}

// // ############## Tests by Raul ############## //
// TEST(DryWetMobility, CanBeCreated){
//   using BD = DryWetBD;
//   BD::Parameters par;
//   par.temperature = 1.0;
//   par.viscosity = 1.0;
//   par.hydrodynamicRadius = 1.0;
//   par.dt = 1.0;
//   par.wetRadius = 0.9;
//   par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
//   par.H = 16;
//   par.Lxy = 32;
//   auto pd = std::make_shared<ParticleData>(1);
//   auto bd = std::make_shared<BD>(pd, par);
// }


// TEST(FullDryMobility, SelfMobilityIsCorrect){
//   using BD = DryWetBD;
//   writeDefaultMobilityFile();
//   BD::Parameters par;
//   par.temperature = 0;
//   par.viscosity = 1.0/(6*M_PI);
//   par.hydrodynamicRadius = 1.0;
//   par.dt = 1.0;
//   par.wetRadius = -1;
//   par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
//   par.dryMobilityFile = "uniformMob.dat";
//   par.H = 32;
//   par.Lxy = 64;
//   auto pd = std::make_shared<ParticleData>(1);
//   pd->getPos(access::cpu, access::write)[0] = real4();
//   auto bd = std::make_shared<BD>(pd, par);
//   bd->addInteractor(std::make_shared<miniInteractor>(pd));
//   bd->forwardTime();
//   real M0 = pd->getPos(access::cpu, access::write)[0].x;
//   ASSERT_THAT(M0, ::testing::DoubleNear(1, 1e-5));
// }

// TEST(FullWetMobility, SelfMobilityIsCorrectAtMiddlePlaneForLargeDomain){
//   using BD = DryWetBD;
//   writeDefaultMobilityFile();
//   BD::Parameters par;
//   par.temperature = 0;
//   par.viscosity = 1.0/(6*M_PI);
//   par.hydrodynamicRadius = 1.0;
//   par.dt = 1.0;
//   par.wetRadius = par.hydrodynamicRadius; //0<wetRadius<=hydrodynamicRadius means full wet
//   par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
//   par.dryMobilityFile = "uniformMob.dat";
//   par.H = 128;
//   par.Lxy = 64;
//   auto pd = std::make_shared<ParticleData>(1);
//   pd->getPos(access::cpu, access::write)[0] = real4();
//   auto bd = std::make_shared<BD>(pd, par);
//   bd->addInteractor(std::make_shared<miniInteractor>(pd));
//   bd->forwardTime();
//   real M0 = pd->getPos(access::cpu, access::write)[0].x;
//   ASSERT_THAT(M0, ::testing::DoubleNear(1, 1e-1));
// }


// //Asserts the correctness of the self mobility for a certain wet radius
// //All other parameters are hardcoded, see the function.
// //The total hydrodynamic radius is 1 (meaning that wetRadius >=1 is full wet)
// void computeSelfMobilityWithWetRadius(real wetRadius){
//   using BD = DryWetBD;
//   writeDefaultMobilityFile();
//   BD::Parameters par;
//   par.temperature = 0;
//   par.viscosity = 1.0/(6*M_PI);
//   par.hydrodynamicRadius = 1.0;
//   par.dt = 1.0;
//   par.wetRadius = wetRadius;
//   par.brownianUpdateRule = DryWetBD::update_rules::euler_maruyama;
//   par.dryMobilityFile = "uniformMob.dat";
//   par.H = 64;
//   par.Lxy = 64;
//   auto pd = std::make_shared<ParticleData>(1);
//   pd->getPos(access::cpu, access::write)[0] = real4();
//   auto bd = std::make_shared<BD>(pd, par);
//   bd->addInteractor(std::make_shared<miniInteractor>(pd));
//   bd->forwardTime();
//   real M0 = pd->getPos(access::cpu, access::write)[0].x;
//   ASSERT_THAT(M0, ::testing::DoubleNear(1, 1e-1))<<"Failed with wet radius "<<wetRadius;
// }

// TEST(DryWetMobility, SelfMobilityIsCorrectForAnyWetRadius){
//   real minWetRadius = 2;
//   real maxWetRadius = 16;
//   int Ntest = 4;
//   fori(0, Ntest){
//     real wetRadius = minWetRadius + i*(maxWetRadius - minWetRadius)/(Ntest-1);
//     computeSelfMobilityWithWetRadius(wetRadius);
//   }
// }
